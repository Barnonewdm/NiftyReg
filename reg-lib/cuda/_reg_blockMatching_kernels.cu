#include "hip/hip_runtime.h"
/*
 *  _reg_blockMatching_kernels.cu
 *
 *
 *  Created by Marc Modat and Pankaj Daga on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef __REG_BLOCKMATCHING_KERNELS_CU__
#define __REG_BLOCKMATCHING_KERNELS_CU__

//#define REDUCE reduceCustom
#define REDUCE blockReduceSum

#include "assert.h"
#include "_reg_blockMatching.h"
// Some parameters that we need for the kernel execution.
// The caller is supposed to ensure that the values are set

// Number of blocks in each dimension
__device__    __constant__ int3 c_BlockDim;
__device__ __constant__ int c_StepSize;
__device__    __constant__ uint3 c_ImageSize;
__device__ __constant__ float r1c1;

// Transformation matrix from nifti header
__device__    __constant__ float4 t_m_a;
__device__    __constant__ float4 t_m_b;
__device__    __constant__ float4 t_m_c;

#define BLOCK_WIDTH 4
#define BLOCK_SIZE 64
#define OVERLAP_SIZE 3
#define STEP_SIZE 1

texture<float, 1, hipReadModeElementType> targetImageArray_texture;
texture<float, 1, hipReadModeElementType> resultImageArray_texture;
texture<int, 1, hipReadModeElementType> activeBlock_texture;

// Apply the transformation matrix
__device__ inline void apply_affine(const float4 &pt, float * result) {
	float4 mat = t_m_a;
	result[0] = (mat.x * pt.x) + (mat.y * pt.y) + (mat.z * pt.z) + (mat.w);
	mat = t_m_b;
	result[1] = (mat.x * pt.x) + (mat.y * pt.y) + (mat.z * pt.z) + (mat.w);
	mat = t_m_c;
	result[2] = (mat.x * pt.x) + (mat.y * pt.y) + (mat.z * pt.z) + (mat.w);
}
template<class DTYPE>
__device__ __inline__
void reg_mat44_mul_cuda(float* mat, DTYPE const* in, DTYPE *out) {
	out[0] = (DTYPE) mat[0 * 4 + 0] * in[0] + (DTYPE) mat[0 * 4 + 1] * in[1] + (DTYPE) mat[0 * 4 + 2] * in[2] + (DTYPE) mat[0 * 4 + 3];
	out[1] = (DTYPE) mat[1 * 4 + 0] * in[0] + (DTYPE) mat[1 * 4 + 1] * in[1] + (DTYPE) mat[1 * 4 + 2] * in[2] + (DTYPE) mat[1 * 4 + 3];
	out[2] = (DTYPE) mat[2 * 4 + 0] * in[0] + (DTYPE) mat[2 * 4 + 1] * in[1] + (DTYPE) mat[2 * 4 + 2] * in[2] + (DTYPE) mat[2 * 4 + 3];
	return;
}

//Marc's kernel
// CUDA kernel to process the target values
__global__ void process_target_blocks_gpu(float *targetPosition_d, float *targetValues) {
	const int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const int3 bDim = c_BlockDim;

	if (tid < bDim.x * bDim.y * bDim.z) {
		const int currentBlockIndex = tex1Dfetch(activeBlock_texture, tid);
		if (currentBlockIndex >= 0) {
			// Get the corresponding (i, j, k) indices
			int tempIndex = currentBlockIndex;
			const int k = (int) (tempIndex / (bDim.x * bDim.y));
			tempIndex -= k * bDim.x * bDim.y;
			const int j = (int) (tempIndex / (bDim.x));
			const int i = tempIndex - j * (bDim.x);
			const int offset = tid * BLOCK_SIZE;
			const int targetIndex_start_x = i * BLOCK_WIDTH;
			const int targetIndex_start_y = j * BLOCK_WIDTH;
			const int targetIndex_start_z = k * BLOCK_WIDTH;

			int targetIndex_end_x = targetIndex_start_x + BLOCK_WIDTH;
			int targetIndex_end_y = targetIndex_start_y + BLOCK_WIDTH;
			int targetIndex_end_z = targetIndex_start_z + BLOCK_WIDTH;
			const uint3 imageSize = c_ImageSize;
			for (int count = 0; count < BLOCK_SIZE; ++count)
				targetValues[count + offset] = 0.0f;
			unsigned int index = 0;

			for (int z = targetIndex_start_z; z < targetIndex_end_z; ++z) {
				if (z >= 0 && z < imageSize.z) {
					int indexZ = z * imageSize.x * imageSize.y;
					for (int y = targetIndex_start_y; y < targetIndex_end_y; ++y) {
						if (y >= 0 && y < imageSize.y) {
							int indexXYZ = indexZ + y * imageSize.x + targetIndex_start_x;
							for (int x = targetIndex_start_x; x < targetIndex_end_x; ++x) {
								if (x >= 0 && x < imageSize.x) {
									targetValues[index + offset] = tex1Dfetch(targetImageArray_texture, indexXYZ);
								}
								indexXYZ++;
								index++;
							}
						} else
							index += BLOCK_WIDTH;
					}
				} else
					index += BLOCK_WIDTH * BLOCK_WIDTH;
			}

			float4 targetPosition;
			targetPosition.x = i * BLOCK_WIDTH;
			targetPosition.y = j * BLOCK_WIDTH;
			targetPosition.z = k * BLOCK_WIDTH;
			apply_affine(targetPosition, &(targetPosition_d[tid * 3]));
		}
	}
}

//Marc's kernel
// CUDA kernel to process the result blocks
__global__ void resultBlocksKernel(float *resultPosition_d, float *targetValues) {

	const int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const int3 bDim = c_BlockDim;
	int tempIndex = tid % NUM_BLOCKS_TO_COMPARE;
	__shared__ int ctid;
	if (tempIndex == 0)
		ctid = (int) (tid / NUM_BLOCKS_TO_COMPARE);
	__syncthreads();
	//const int ctid = (int)(tid / NUM_BLOCKS_TO_COMPARE);
	__shared__ float4 localCC[NUM_BLOCKS_TO_COMPARE];
	__shared__ int3 indexes;
	localCC[tempIndex] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	__shared__ int updateThreadID;
	updateThreadID = -1;
	if (ctid < bDim.x * bDim.y * bDim.z) {
		const int activeBlockIndex = tex1Dfetch(activeBlock_texture, ctid);
		tempIndex = activeBlockIndex;
		int k = (int) (tempIndex / (bDim.x * bDim.y));
		tempIndex -= k * bDim.x * bDim.y;
		int j = (int) (tempIndex / (bDim.x));
		int i = tempIndex - j * (bDim.x);
		tempIndex = tid % NUM_BLOCKS_TO_COMPARE;
		if (tempIndex == 0) {
			indexes.x = i * BLOCK_WIDTH;
			indexes.y = j * BLOCK_WIDTH;
			indexes.z = k * BLOCK_WIDTH;
		}
		__syncthreads();

		if (activeBlockIndex >= 0) {
			const int block_offset = ctid * BLOCK_SIZE;
			const uint3 imageSize = c_ImageSize;
			int k = (int) tempIndex / NUM_BLOCKS_TO_COMPARE_2D;
			tempIndex -= k * NUM_BLOCKS_TO_COMPARE_2D;
			int j = (int) tempIndex / NUM_BLOCKS_TO_COMPARE_1D;
			int i = tempIndex - j * NUM_BLOCKS_TO_COMPARE_1D;
			k -= OVERLAP_SIZE;
			j -= OVERLAP_SIZE;
			i -= OVERLAP_SIZE;
			tempIndex = tid % NUM_BLOCKS_TO_COMPARE;
			int resultIndex_start_z = indexes.z + k;
			int resultIndex_end_z = resultIndex_start_z + BLOCK_WIDTH;
			int resultIndex_start_y = indexes.y + j;
			int resultIndex_end_y = resultIndex_start_y + BLOCK_WIDTH;
			int resultIndex_start_x = indexes.x + i;
			int resultIndex_end_x = resultIndex_start_x + BLOCK_WIDTH;
			__shared__ float4 cc_vars[NUM_BLOCKS_TO_COMPARE];
			cc_vars[tempIndex].x = 0.0f;
			cc_vars[tempIndex].y = 0.0f;
			unsigned int index = 0;
			for (int z = resultIndex_start_z; z < resultIndex_end_z; ++z) {
				if (z >= 0 && z < imageSize.z) {
					int indexZ = z * imageSize.y * imageSize.x;
					for (int y = resultIndex_start_y; y < resultIndex_end_y; ++y) {
						if (y >= 0 && y < imageSize.y) {
							int indexXYZ = indexZ + y * imageSize.x + resultIndex_start_x;
							for (int x = resultIndex_start_x; x < resultIndex_end_x; ++x) {
								if (x >= 0 && x < imageSize.x) {
									cc_vars[tempIndex].x = tex1Dfetch(resultImageArray_texture, indexXYZ);
									cc_vars[tempIndex].y = targetValues[block_offset + index];
									if (cc_vars[tempIndex].x != 0.0f && cc_vars[tempIndex].y != 0.0f) {
										localCC[tempIndex].x += cc_vars[tempIndex].x;
										localCC[tempIndex].y += cc_vars[tempIndex].y;
										++localCC[tempIndex].z;
									}
								}
								++indexXYZ;
								++index;
							}
						} else
							index += BLOCK_WIDTH;
					}
				} else
					index += BLOCK_WIDTH * BLOCK_WIDTH;
			}

			if (localCC[tempIndex].z > 0) {
				localCC[tempIndex].x /= localCC[tempIndex].z;
				localCC[tempIndex].y /= localCC[tempIndex].z;
			}
			cc_vars[tempIndex].z = 0.0f;
			cc_vars[tempIndex].w = 0.0f;
			index = 0;
			for (int z = resultIndex_start_z; z < resultIndex_end_z; ++z) {
				if (z >= 0 && z < imageSize.z) {
					int indexZ = z * imageSize.y * imageSize.x;
					for (int y = resultIndex_start_y; y < resultIndex_end_y; ++y) {
						if (y >= 0 && y < imageSize.y) {
							int indexXYZ = indexZ + y * imageSize.x + resultIndex_start_x;
							for (int x = resultIndex_start_x; x < resultIndex_end_x; ++x) {
								if (x >= 0 && x < imageSize.x) {
									cc_vars[tempIndex].x = tex1Dfetch(resultImageArray_texture, indexXYZ);
									cc_vars[tempIndex].y = targetValues[block_offset + index];
									if (cc_vars[tempIndex].x != 0.0f && cc_vars[tempIndex].y != 0.0f) {
										cc_vars[tempIndex].x -= localCC[tempIndex].x;
										cc_vars[tempIndex].y -= localCC[tempIndex].y;

										cc_vars[tempIndex].z += cc_vars[tempIndex].x * cc_vars[tempIndex].x;
										cc_vars[tempIndex].w += cc_vars[tempIndex].y * cc_vars[tempIndex].y;
										localCC[tempIndex].w += cc_vars[tempIndex].x * cc_vars[tempIndex].y;
									}
								}
								++indexXYZ;
								++index;
							}
						} else
							index += BLOCK_WIDTH;
					}
				} else
					index += BLOCK_WIDTH * BLOCK_WIDTH;
			}

			if (localCC[tempIndex].z > (float) (BLOCK_SIZE / 2)) {
				if (cc_vars[tempIndex].z > 0.0f && cc_vars[tempIndex].w > 0.0f) {
					localCC[tempIndex].w = fabsf(localCC[tempIndex].w / sqrt(cc_vars[tempIndex].z * cc_vars[tempIndex].w));
				}
			} else {
				localCC[tempIndex].w = 0.0f;
			}

			localCC[tempIndex].x = i;
			localCC[tempIndex].y = j;
			localCC[tempIndex].z = k;

			// Just take ownership of updating the final value
			if (updateThreadID == -1)
				updateThreadID = tid;
		}
		__syncthreads();

		// Just let one thread do the final update
		if (tid == updateThreadID) {
			__shared__ float4 bestCC;
			bestCC = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			for (int i = 0; i < NUM_BLOCKS_TO_COMPARE; ++i) {
				if (localCC[i].w > bestCC.w) {
					bestCC.x = localCC[i].x;
					bestCC.y = localCC[i].y;
					bestCC.z = localCC[i].z;
					bestCC.w = localCC[i].w;
				}
			}
			bestCC.x += indexes.x;
			bestCC.y += indexes.y;
			bestCC.z += indexes.z;
			apply_affine(bestCC, &(resultPosition_d[ctid * 3]));
		}
	}
}

__device__ __inline__ void reduceCC(float* sData, const unsigned int tid, const unsigned int blockSize) {

	if (blockSize >= 512) {
		if (tid < 256) {
			sData[tid] += sData[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sData[tid] += sData[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sData[tid] += sData[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if (blockSize >= 64)
			sData[tid] += sData[tid + 32];
		if (blockSize >= 32)
			sData[tid] += sData[tid + 16];
		if (blockSize >= 16)
			sData[tid] += sData[tid + 8];
		if (blockSize >= 8)
			sData[tid] += sData[tid + 4];
		if (blockSize >= 4)
			sData[tid] += sData[tid + 2];
		if (blockSize >= 2)
			sData[tid] += sData[tid + 1];
	}
}

__device__ __inline__ void reduce(float* sData, const unsigned int tid, const unsigned int blockSize) {

	if (blockSize >= 512) {
		if (tid < 256) {
			sData[tid] += sData[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sData[tid] += sData[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sData[tid] += sData[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if (blockSize >= 64)
			sData[tid] += sData[tid + 32];
		if (blockSize >= 32)
			sData[tid] += sData[tid + 16];
		if (blockSize >= 16)
			sData[tid] += sData[tid + 8];
		if (blockSize >= 8)
			sData[tid] += sData[tid + 4];
		if (blockSize >= 4)
			sData[tid] += sData[tid + 2];
		if (blockSize >= 2)
			sData[tid] += sData[tid + 1];
	}
}

//must parameterize warpsize in both cuda and cl
__device__ __inline__ float reduceCustom_f1(float data, const unsigned int tid, const unsigned int blockSize) {
	static __shared__ float sDataBuff[8 * 8 * 8];

	sDataBuff[tid] = data;
	__syncthreads();

	const unsigned int warpId = tid / 32;
	const unsigned int bid = tid / blockSize;

	if (warpId % 2 == 0) {
		sDataBuff[tid] += sDataBuff[tid + 32];
		sDataBuff[tid] += sDataBuff[tid + 16];
		sDataBuff[tid] += sDataBuff[tid + 8];
		sDataBuff[tid] += sDataBuff[tid + 4];
		sDataBuff[tid] += sDataBuff[tid + 2];
		sDataBuff[tid] += sDataBuff[tid + 1];
	}

	__syncthreads();
	return sDataBuff[bid * blockSize];
}

__device__ __inline__ float reduceCustom_f(float data, const unsigned int tid) {
	static __shared__ float sData2[64];

	sData2[tid] = data;
	__syncthreads();

	if (tid < 32) {
		sData2[tid] += sData2[tid + 32];
		sData2[tid] += sData2[tid + 16];
		sData2[tid] += sData2[tid + 8];
		sData2[tid] += sData2[tid + 4];
		sData2[tid] += sData2[tid + 2];
		sData2[tid] += sData2[tid + 1];
	}

	__syncthreads();
	return sData2[0];
}

__device__ __inline__ float reduceCustom(float data, const unsigned int tid) {
	static __shared__ float sData2[64];

	sData2[tid] = data;
	__syncthreads();

	if (tid < 32)
		sData2[tid] += sData2[tid + 32];
	if (tid < 16)
		sData2[tid] += sData2[tid + 16];
	if (tid < 8)
		sData2[tid] += sData2[tid + 8];
	if (tid < 4)
		sData2[tid] += sData2[tid + 4];
	if (tid < 2)
		sData2[tid] += sData2[tid + 2];
	if (tid == 0)
		sData2[0] += sData2[1];

	__syncthreads();
	return sData2[0];
}

__inline__ __device__
float warpAllReduceSum(float val) {
	for (int mask = 16; mask > 0; mask /= 2)
		val += __shfl_xor(val, mask);
	return val;
}

__inline__ __device__
float warpReduceSum(float val) {
	for (int offset = 16; offset > 0; offset /= 2)
		val += __shfl_down(val, offset);
	return val;
}

__inline__ __device__
float blockReduceSum(float val, int tid) {

	static __shared__ float shared[2];
	int laneId = tid % 32;
	int warpId = tid / 32;

	val = warpReduceSum(val);     // Each warp performs partial reduction

	if (laneId == 0)
		shared[warpId] = val;
	//if (blockIdx.x == 8 && blockIdx.y == 0 && blockIdx.z == 0) printf("idx: %d | lane: %d \n", tid, lane);
	__syncthreads();

	return shared[0] + shared[1];
}

__device__ __inline__ void rewind(float* sValues, unsigned int tid) {

	while (tid < 11 * 11 * 11) {
		const float buffer = sValues[tid];
		__syncthreads();
		if (tid > 0)
			sValues[tid - 1] = buffer;

		tid += blockDim.x;
	}
}

__global__ void blockMatchingKernel(float *resultPosition, float *targetPosition, int* mask, float* targetMatrix_xyz, unsigned int* definedBlock, uint3 c_ImageSize) {

	__shared__ float sResultValues[12 * 12 * 12];

	const unsigned int idz = threadIdx.x / 16;
	const unsigned int idy = (threadIdx.x - 16 * idz) / 4;
	const unsigned int idx = threadIdx.x - 16 * idz - 4 * idy;

	const unsigned int blockIndex = blockIdx.x + gridDim.x * blockIdx.y + (gridDim.x * gridDim.y) * blockIdx.z;

	const unsigned int xBaseImage = blockIdx.x * 4;
	const unsigned int yBaseImage = blockIdx.y * 4;
	const unsigned int zBaseImage = blockIdx.z * 4;

//	bool predicate = xBaseImage == 16 && yBaseImage == 24 && zBaseImage == 24;

	const unsigned int tid = threadIdx.x;     //0-blockSize

	const unsigned int xImage = xBaseImage + idx;
	const unsigned int yImage = yBaseImage + idy;
	const unsigned int zImage = zBaseImage + idz;

	const unsigned long imgIdx = xImage + yImage * (c_ImageSize.x) + zImage * (c_ImageSize.x * c_ImageSize.y);
	const bool targetInBounds = xImage < c_ImageSize.x && yImage < c_ImageSize.y && zImage < c_ImageSize.z;

	const int currentBlockIndex = tex1Dfetch(activeBlock_texture, blockIndex);

	if (currentBlockIndex > -1) {

		float bestDisplacement[3] = { nanf("sNaN"), 0.0f, 0.0f };
		float bestCC = 0.0f;

		//populate shared memory with resultImageArray's values
		for (int n = -1; n <= 1; n += 1) {
			for (int m = -1; m <= 1; m += 1) {
				for (int l = -1; l <= 1; l += 1) {
					const int x = l * 4 + idx;
					const int y = m * 4 + idy;
					const int z = n * 4 + idz;

					const unsigned int sIdx = (z + 4) * 12 * 12 + (y + 4) * 12 + (x + 4);

					const int xImageIn = xBaseImage + x;
					const int yImageIn = yBaseImage + y;
					const int zImageIn = zBaseImage + z;

					const int indexXYZIn = xImageIn + yImageIn * (c_ImageSize.x) + zImageIn * (c_ImageSize.x * c_ImageSize.y);

					const bool valid = (xImageIn >= 0 && xImageIn < c_ImageSize.x) && (yImageIn >= 0 && yImageIn < c_ImageSize.y) && (zImageIn >= 0 && zImageIn < c_ImageSize.z);
					sResultValues[sIdx] = (valid /*&& mask[indexXYZIn]>-1*/) ? tex1Dfetch(resultImageArray_texture, indexXYZIn) : nanf("sNaN");

				}
			}
		}

		//for most cases we need this out of th loop
		//value if the block is 4x4x4 NaN otherwise
		float rTargetValue = (targetInBounds /*&& mask[imgIdx]>-1*/) ? tex1Dfetch(targetImageArray_texture, imgIdx) : nanf("sNaN");
		const bool finiteTargetIntensity = isfinite(rTargetValue);
		rTargetValue = finiteTargetIntensity ? rTargetValue : 0.f;

		const unsigned int targetBlockSize = __syncthreads_count(finiteTargetIntensity);

		if (targetBlockSize > 32) {
			//the target values must remain constant throughout the block matching process
			const float targetMean = __fdividef(REDUCE(rTargetValue, tid) , targetBlockSize);
			const float targetTemp = finiteTargetIntensity ? rTargetValue - targetMean : 0.f;
			const float targetVar = REDUCE(targetTemp * targetTemp, tid);

			// iteration over the result blocks (block matching part)
			for (unsigned int n = 1; n < 8; n += 1) {
				for (unsigned int m = 1; m < 8; m += 1) {
					for (unsigned int l = 1; l < 8; l += 1) {

						const unsigned int sIdxIn = (idz + n) * 144 /*12*12*/+ (idy + m) * 12 + idx + l;
						const float rResultValue = sResultValues[sIdxIn];
						const bool overlap = isfinite(rResultValue) && finiteTargetIntensity;
						const unsigned int blockSize = __syncthreads_count(overlap);

						if (blockSize > 32) {

							//the target values must remain constant at each loop, so please do not touch this!
							float newTargetTemp = targetTemp;
							float newTargetVar = targetVar;
							if (blockSize != targetBlockSize) {

								const float newTargetValue = overlap ? rTargetValue : 0.0f;
								const float newTargetMean = __fdividef(REDUCE(newTargetValue, tid) , blockSize);
								newTargetTemp = overlap ? newTargetValue - newTargetMean : 0.0f;
								newTargetVar = REDUCE(newTargetTemp * newTargetTemp, tid);
							}

							const float rChecked = overlap ? rResultValue : 0.0f;
							const float resultMean = __fdividef(REDUCE(rChecked, tid),blockSize)  ;
							const float resultTemp = overlap ? rChecked - resultMean : 0.0f;
							const float resultVar = REDUCE(resultTemp * resultTemp, tid);

							const float sumTargetResult =  REDUCE((newTargetTemp) * (resultTemp), tid);
							const float localCC = fabs((sumTargetResult) * rsqrtf( newTargetVar *  resultVar));

//							if (predicate && tid==0 && 0.981295-localCC<0.04 && fabs(0.981295 - localCC)>=0) printf("G|%d-%d-%d|%d|TMN:%f|TVR:%f|RMN:%f|RVR:%f|LCC:%lf|BCC:%lf\n",l-4,m-4,n-4, blockSize, targetMean, targetVar, resultMean, resultVar, localCC, bestCC);

							//temporary for testing
//							if (tid == 0 && bestCC != 0.981295f && (localCC > bestCC )) {
							if (tid == 0 && localCC > bestCC ) {
								bestCC = localCC;
								bestDisplacement[0] = l - 4.0f;
								bestDisplacement[1] = m - 4.0f;
								bestDisplacement[2] = n - 4.0f;
							}
							/*if (predicate && tid==0 )
								printf("C|%d-%d-%d|%f-%f-%f\n", l-4, m-4, n-4, bestDisplacement[0], bestDisplacement[1], bestDisplacement[2]);*/
						}
					}
				}
			}

			if (tid == 0 && isfinite(bestDisplacement[0])) {
				const unsigned int posIdx = 3 * atomicAdd(definedBlock, 1);
//				if(predicate)printf("defined: %d | %d-%d-%d\n", *definedBlock, xBaseImage, yBaseImage, zBaseImage);

				resultPosition += posIdx;
				targetPosition += posIdx;

				const float targetPosition_temp[3] = { xBaseImage, yBaseImage, zBaseImage };

				bestDisplacement[0] += targetPosition_temp[0];
				bestDisplacement[1] += targetPosition_temp[1];
				bestDisplacement[2] += targetPosition_temp[2];

				//float  tempPosition[3];
				reg_mat44_mul_cuda<float>(targetMatrix_xyz, targetPosition_temp, targetPosition);
				reg_mat44_mul_cuda<float>(targetMatrix_xyz, bestDisplacement, resultPosition);
			}
		}
	}

}

#endif
