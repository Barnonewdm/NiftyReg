#include "hip/hip_runtime.h"
/*
 *  _reg_blockMatching_gpu.cu
 *
 *
 *  Created by Marc Modat and Pankaj Daga on 24/03/2009.
 *  Copyright 2009 UCL - CMIC. All rights reserved.
 *
 */

#ifndef _REG_BLOCKMATCHING_GPU_CU
#define _REG_BLOCKMATCHING_GPU_CU

#include "_reg_blockMatching_gpu.h"
#include "_reg_blockMatching_kernels.cu"

//#include "_reg_blocksize_gpu.h"
#include "_reg_ReadWriteImage.h"
#include "_reg_tools.h"
#include "hipblas.h"
#include "hipsolver.h"

#include <vector>
#include "_reg_maths.h"

#include "CudaKernelFuncs.h"

#include "nvToolsExt.h"
#include "nvToolsExtCuda.h"

/* *************************************************************** */

void block_matching_method_gpu(nifti_image *targetImage, _reg_blockMatchingParam *params, float **targetImageArray_d, float **resultImageArray_d, float **targetPosition_d, float **resultPosition_d, int **activeBlock_d, int **mask_d, float** targetMat_d) {

	// Copy some required parameters over to the device
	uint3 imageSize = make_uint3(targetImage->nx, targetImage->ny, targetImage->nz); // Image size

	// Texture binding
	const unsigned int numBlocks = params->blockNumber[0] * params->blockNumber[1] * params->blockNumber[2];
	NR_CUDA_SAFE_CALL(hipBindTexture(0, targetImageArray_texture, *targetImageArray_d, targetImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipBindTexture(0, resultImageArray_texture, *resultImageArray_d, targetImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipBindTexture(0, activeBlock_texture, *activeBlock_d, numBlocks * sizeof(int)));

	unsigned int* definedBlock_d;
	unsigned int *definedBlock_h = (unsigned int*) malloc(sizeof(unsigned int));
	*definedBlock_h = 0;
	NR_CUDA_SAFE_CALL(hipMalloc((void** )(&definedBlock_d), sizeof(unsigned int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(definedBlock_d, definedBlock_h, sizeof(unsigned int), hipMemcpyHostToDevice));

	dim3 BlockDims1D(64, 1, 1);
	dim3 BlocksGrid3D(params->blockNumber[0], params->blockNumber[1], params->blockNumber[2]);
	const int blockRange = params->voxelCaptureRange % 4 ? params->voxelCaptureRange / 4 + 1 : params->voxelCaptureRange / 4;
	const unsigned int sMem = (blockRange * 2 + 1) * (blockRange * 2 + 1) * (blockRange * 2 + 1) * 64 * sizeof(float);
	blockMatchingKernel<< <BlocksGrid3D, BlockDims1D, sMem >> >(*resultPosition_d, *targetPosition_d, *mask_d, *targetMat_d, definedBlock_d, imageSize, blockRange, params->stepSize);

#ifndef NDEBUG
	NR_CUDA_CHECK_KERNEL(BlocksGrid3D, BlockDims1D)
#endif
	NR_CUDA_SAFE_CALL(hipDeviceSynchronize());

	NR_CUDA_SAFE_CALL(hipMemcpy((void * )definedBlock_h, (void * )definedBlock_d, sizeof(unsigned int), hipMemcpyDeviceToHost));
	params->definedActiveBlock = *definedBlock_h;
//	printf("kernel definedActiveBlock: %d\n", params->definedActiveBlock);
	NR_CUDA_SAFE_CALL(hipUnbindTexture(targetImageArray_texture));
	NR_CUDA_SAFE_CALL(hipUnbindTexture(resultImageArray_texture));
	NR_CUDA_SAFE_CALL(hipUnbindTexture(activeBlock_texture));

	free(definedBlock_h);
	hipFree(definedBlock_d);

}

//------------Optimizer------------------------------------

void checkCublasStatus(hipblasStatus_t status) {
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! CUBLAS  error\n");
		exit(0);
	}
}
void checkCUSOLVERStatus(hipsolverStatus_t status, char* msg) {
	if (status != HIPSOLVER_STATUS_SUCCESS) {
		if (status == HIPSOLVER_STATUS_SUCCESS)
			printf("%s: PASS\n", msg);
		else if (status == HIPSOLVER_STATUS_NOT_INITIALIZED)
			printf("%s: the library was not initialized.\n", msg);
		else if (status == HIPSOLVER_STATUS_INVALID_VALUE)
			printf("%s: invalid parameters were passed (m,n<0 or lda<max(1,m) or ldu<max(1,m) or ldvt<max(1,n) ).\n", msg);
		else if (status == HIPSOLVER_STATUS_ARCH_MISMATCH)
			printf("%s: the device only supports compute capability 2.0 and above.\n", msg);
		else if (status == HIPSOLVER_STATUS_INTERNAL_ERROR)
			printf("%s: an internal operation failed.\n", msg);
		else if (status == HIPSOLVER_STATUS_EXECUTION_FAILED)
			printf("%s: a kernel failed to launch on the GPU.\n", msg);
		else
			printf("%s: %d\n", msg, status);
		exit(0);
	}
}
void checkDevInfo(int *devInfo) {
	int * hostDevInfo = (int*) malloc(sizeof(int));
	hipMemcpy(hostDevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	if (hostDevInfo < 0)
		printf("parameter: %d is wrong\n", hostDevInfo);
	if (hostDevInfo > 0)
		printf("%d superdiagonals of an intermediate bidiagonal form B did not converge to zero.\n", hostDevInfo);
	else
		printf(" %d: operation successful\n", hostDevInfo);
	free(hostDevInfo);
}
void downloadMat44(mat44 *lastTransformation, float* transform_d) {
	float* tempMat = (float*) malloc(16 * sizeof(float));
	hipMemcpy(tempMat, transform_d, 16 * sizeof(float), hipMemcpyDeviceToHost);
	cPtrToMat44(lastTransformation, tempMat);
	free(tempMat);
}
void uploadMat44(mat44 lastTransformation, float* transform_d) {
	float* tempMat = (float*) malloc(16 * sizeof(float));
	mat44ToCptr(lastTransformation, tempMat);
	hipMemcpy(transform_d, tempMat, 16 * sizeof(float), hipMemcpyHostToDevice);
	free(tempMat);
}
/*
 * the function computes the SVD of a matrix A
 * A = V* x S x U, where V* is a (conjugate) transpose of V
 * */
void cusolverSVD(float* A_d, unsigned int m, unsigned int n, float* S_d, float* VT_d, float* U_d) {

	const int lda = m;
	const int ldu = m;
	const int ldvt = n;

	/*
	 * 'A': all m columns of U are returned in array
	 * 'S': the first min(m,n) columns of U (the left singular vectors) are returned in the array
	 * 'O': the first min(m,n) columns of U (the left singular vectors) are overwritten on the array
	 * 'N': no columns of U (no left singular vectors) are computed
	 */
	const char jobu = 'A';

	/*
	 * 'A': all N rows of V**T are returned in the array
	 * 'S': the first min(m,n) rows of V**T (the right singular vectors) are returned in the array
	 * 'O': the first min(m,n) rows of V**T (the right singular vectors) are overwritten on the array
	 * 'N': no rows of V**T (no right singular vectors) are computed
	 */
	const char jobvt = 'A';

	hipsolverHandle_t gH = NULL;
	int Lwork;
	//device ptrs
	float *Work;
	float *rwork;
	int *devInfo;
	nvtxNameOsThread(1, "MAIN");
	nvtxRangePush(__FUNCTION__);

	//init cusolver compute SVD and shut down
	nvtxMark("Init...");
	checkCUSOLVERStatus(hipsolverDnCreate(&gH), "hipsolverDnCreate");
	nvtxMark("Lwork...");
	checkCUSOLVERStatus(hipsolverDnSgesvd_bufferSize(gH, m, n, &Lwork), "hipsolverDnSgesvd_bufferSize");

	hipMalloc(&Work, Lwork * sizeof(float));
	hipMalloc(&rwork, Lwork * sizeof(float));
	hipMalloc(&devInfo, sizeof(int));
	nvtxMark("SVD...");
	checkCUSOLVERStatus(hipsolverDnSgesvd(gH, jobu, jobvt, m, n, A_d, lda, S_d, U_d, ldu, VT_d, ldvt, Work, Lwork, NULL, devInfo), "hipsolverDnSgesvd");
	nvtxMark("Destroy!!!");
	checkCUSOLVERStatus(hipsolverDnDestroy(gH), "hipsolverDnDestroy");
	nvtxRangePop();
	//free vars
	hipFree(devInfo);
	hipFree(rwork);
	hipFree(Work);

}
/*
 * the function computes the Pseudoinverse from the products of the SVD factorisation of A
 * R = V x inv(S) x U*
 * */
void cublasPseudoInverse(float* transformation, float *R_d, float* result_d, float *VT_d, float* Sigma_d, float *U_d, const unsigned int m, const unsigned int n) {
	// First we make sure that the really small singular values
	// are set to 0. and compute the inverse by taking the reciprocal of the entries
	nvtxNameOsThread(1, "MAIN");
	nvtxRangePush(__FUNCTION__);
	trimAndInvertSingularValuesKernel<<<1, n>>>(Sigma_d);	//test 3

	hipblasHandle_t handle;

	const float alpha = 1.f;
	const float beta = 0.f;

	const int ldvt = n;	//VT's lead dimension
	const int ldu = m;	//U's lead dimension
	const int ldr = n;	//Pseudoinverse's r lead dimension

	const int rowsVTandR = n;	//VT and r's num rows
	const int colsUandR = m;	//U and r's num cols
	const int colsVtRowsU = n;	//VT's cols and U's rows

	// V x inv(S) in place | We scale eaach row with the corresponding singular value as V is transpose
	scaleV<<<n,n>>>(VT_d, n, n, Sigma_d);

	//Initialize CUBLAS perform ops and shut down
	checkCublasStatus(hipblasCreate(&handle));

	//now R = V x inv(S) x U*
	checkCublasStatus(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, rowsVTandR, colsUandR, colsVtRowsU, &alpha, VT_d, ldvt, U_d, ldu, &beta, R_d, ldr));

	//finally M=Rxb, where M is our affine matrix and b a vector containg the result points
	checkCublasStatus(hipblasSgemv(handle, HIPBLAS_OP_N, n, m, &alpha, R_d, ldr, result_d, 1, &beta, transformation, 1));
	checkCublasStatus(hipblasDestroy(handle));
	permuteAffineMatrix<<<1,16>>>(transformation);
	hipDeviceSynchronize();
	nvtxRangePop();

}

//OPTIMIZER-----------------------------------------------

// estimate an affine transformation using least square
void getAffineMat3D(float* AR_d, float* Sigma_d, float* VT_d, float* U_d, float* target_d, float* result_d, float *transformation, const unsigned int numBlocks, unsigned int m, unsigned int n) {

//populate A
	populateMatrixA<<<numBlocks, 512>>>(AR_d,target_d, m/3); //test 2

//calculate SVD on the GPU

	nvtxNameOsThread(1,"MAIN");
	nvtxRangePush(__FUNCTION__);
	cusolverSVD(AR_d, m, n, Sigma_d, VT_d, U_d);

	cublasPseudoInverse(transformation, AR_d,result_d, VT_d,Sigma_d, U_d, m, n);
	nvtxRangePop();

}

void optimize_affine3D_cuda(mat44* cpuMat, float* final_d, float* AR_d, float* U_d, float* Sigma_d, float* VT_d, float* lengths_d, float* target_d, float* result_d, float* newResult_d, unsigned int m, unsigned int n, const unsigned int numToKeep, bool ilsIn) {

	//m | blockMatchingParams->definedActiveBlock * 3
	//n | 12
	const unsigned int numEquations = m / 3;
	const unsigned int numBlocks = (numEquations % 512) ? (numEquations / 512) + 1 : numEquations / 512;

	uploadMat44(*cpuMat, final_d);
	transformResultPointsKernel<<<numBlocks, 512>>>(final_d, result_d,newResult_d, m/3); //test 1
	hipMemcpy(result_d, newResult_d, m * sizeof(float), hipMemcpyDeviceToDevice);

	// run the local search optimization routine
	affineLocalSearch3DCuda(cpuMat, final_d, AR_d, Sigma_d, U_d, VT_d, newResult_d, target_d, result_d, lengths_d, numBlocks, numToKeep, m, n);

	downloadMat44(cpuMat, final_d);
}
void affineLocalSearch3DCuda(mat44 *cpuMat, float* final_d, float *AR_d, float* Sigma_d, float* U_d, float* VT_d, float * newResultPos_d, float* targetPos_d, float* resultPos_d, float* lengths_d, const unsigned int numBlocks, const unsigned int num_to_keep, const unsigned int m, const unsigned int n) {

	double lastDistance = std::numeric_limits<double>::max();

	float* lastTransformation_d;
	hipMalloc(&lastTransformation_d, 16 * sizeof(float));

	//get initial affine matrix
	getAffineMat3D(AR_d, Sigma_d, VT_d, U_d, targetPos_d, resultPos_d, final_d, numBlocks, m, n);

	for (unsigned int count = 0; count < MAX_ITERATIONS; ++count) {

		// Transform the points in the target
		transformResultPointsKernel<<<numBlocks, 512>>>(final_d, targetPos_d,newResultPos_d, m/3); //test 1
		double distance = sortAndReduce( lengths_d, targetPos_d, resultPos_d, newResultPos_d, numBlocks,num_to_keep, m);

		// If the change is not substantial or we are getting worst, we return
		if ((distance >= lastDistance) || (lastDistance - distance) < TOLERANCE) break;

		lastDistance = distance;

		hipMemcpy( lastTransformation_d,final_d, 16*sizeof(float), hipMemcpyDeviceToDevice);
		getAffineMat3D(AR_d, Sigma_d, VT_d, U_d, targetPos_d, resultPos_d, final_d, numBlocks, m, n);
	}

	//async cudamemcpy here
	hipMemcpy(final_d, lastTransformation_d, 16 * sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(lastTransformation_d);
}

#endif
