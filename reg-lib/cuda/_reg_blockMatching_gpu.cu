#include "hip/hip_runtime.h"
/*
 *  _reg_blockMatching_gpu.cu
 *
 *
 *  Created by Marc Modat and Pankaj Daga on 24/03/2009.
 *  Copyright 2009 UCL - CMIC. All rights reserved.
 *
 */

#ifndef _REG_BLOCKMATCHING_GPU_CU
#define _REG_BLOCKMATCHING_GPU_CU

#include "_reg_blockMatching_gpu.h"
#include "_reg_blockMatching_kernels.cu"

#include "_reg_blocksize_gpu.h"
#include "_reg_ReadWriteImage.h"
#include "_reg_tools.h"




void block_matching_method_gpu3(nifti_image *targetImage, _reg_blockMatchingParam *params, float **targetImageArray_d, float **resultImageArray_d, float **targetPosition_d, float **resultPosition_d, int **activeBlock_d, int **mask_d) {

	// Copy some required parameters over to the device
	int3 bDim = make_int3(params->blockNumber[0], params->blockNumber[1], params->blockNumber[2]);
	uint3 image_size = make_uint3(targetImage->nx, targetImage->ny, targetImage->nz);// Image size
	NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_BlockDim), &bDim, sizeof(uint3)));
	NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize), &image_size, sizeof(uint3)));

	// Texture binding
	const unsigned int numBlocks = params->blockNumber[0] * params->blockNumber[1] * params->blockNumber[2];
	NR_CUDA_SAFE_CALL(hipBindTexture(0, targetImageArray_texture, *targetImageArray_d, targetImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipBindTexture(0, resultImageArray_texture, *resultImageArray_d, targetImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipBindTexture(0, activeBlock_texture, *activeBlock_d, numBlocks * sizeof(int)));

	mat44 targetMatrix_xyz = (targetImage->sform_code > 0) ? (targetImage->sto_xyz) : (targetImage->qto_xyz);
	float* targetMat = (float *)malloc(16 * sizeof(float));//freed
	mat44ToCptr(targetMatrix_xyz, targetMat);

	float* targetMat_d;//freed
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&targetMat_d), 16 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(targetMat_d, targetMat, 16 * sizeof(float), hipMemcpyHostToDevice));

	unsigned int* definedBlock_d;
	unsigned int *definedBlock_h = (unsigned int*)malloc(sizeof(unsigned int));
	definedBlock_h[0] = 0;
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&definedBlock_d), sizeof(unsigned int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(definedBlock_d, definedBlock_h, sizeof(unsigned int), hipMemcpyHostToDevice));



	dim3 BlockDims1D(64, 1, 1);
	dim3 BlocksGrid3D(params->blockNumber[0], params->blockNumber[1], params->blockNumber[2]);
	const uint3 blockSize = make_uint3(4, 4, 4);


	blockMatchingKernel << <BlocksGrid3D, BlockDims1D >> >(*resultPosition_d, *targetPosition_d, *mask_d, targetMat_d, blockSize, definedBlock_d);
	//NR_CUDA_CHECK_KERNEL(BlocksGrid3D, BlockDims1D)

	NR_CUDA_SAFE_CALL(hipDeviceSynchronize());

	NR_CUDA_SAFE_CALL(hipMemcpy((void *)definedBlock_h, (void *)definedBlock_d, sizeof(unsigned int), hipMemcpyDeviceToHost));
	params->definedActiveBlock = definedBlock_h[0];
	//printf("definedActiveBlock: %d\n", params->definedActiveBlock);
	NR_CUDA_SAFE_CALL(hipUnbindTexture(targetImageArray_texture));
	NR_CUDA_SAFE_CALL(hipUnbindTexture(resultImageArray_texture));
	NR_CUDA_SAFE_CALL(hipUnbindTexture(activeBlock_texture));

	free(definedBlock_h);
	free(targetMat);
	hipFree(targetMat_d);
	hipFree(definedBlock_d);

}


void optimize_gpu(_reg_blockMatchingParam *blockMatchingParams,
	mat44 *updateAffineMatrix,
	float **targetPosition_d,
	float **resultPosition_d,
	bool affine)
{

	// Cheat and call the CPU version.
	optimize(blockMatchingParams, updateAffineMatrix, affine);

}

#endif
