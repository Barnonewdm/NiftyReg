#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include"_reg_blocksize_gpu.h"
#include"_reg_resampling.h"
#include"_reg_maths.h"
#include "cudaKernelFuncs.h"
#include "_reg_common_gpu.h"

#include"_reg_tools.h"
#include"_reg_ReadWriteImage.h"
#include "hip/hip_runtime_api.h"


#include "_reg_resampling.h"
#include "_reg_maths.h"
#include "_reg_blockMatching_gpu.h"
#include "_reg_blockMatching.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

unsigned int min1(unsigned int a, unsigned int b) {
	return (a < b) ? a : b;
}

texture<float, 3, hipReadModeElementType> floatingTexture;

__device__ __constant__ float cIdentity[16];
void runKernel(nifti_image *floatingImage, nifti_image *warpedImage, nifti_image *deformationFieldImage, int *mask, int interp, float paddingValue, int *dtiIndeces, mat33 * jacMat);

__device__ __inline__ void reg_mat44_expm_cuda(float* mat) {
	//todo 
}

__device__ __inline__
void reg_mat44_logm_cuda(float* mat) {
	//todo
}


template <class DTYPE>
__device__ __inline__ void reg_mat44_mul_cuda(DTYPE const* mat, DTYPE const* in, DTYPE *out) {
	out[0] = mat[0 * 4 + 0] * in[0] +
		mat[0 * 4 + 1] * in[1] +
		mat[0 * 4 + 2] * in[2] +
		mat[0 * 4 + 3];
	out[1] = mat[1 * 4 + 0] * in[0] +
		mat[1 * 4 + 1] * in[1] +
		mat[1 * 4 + 2] * in[2] +
		mat[1 * 4 + 3];
	out[2] = mat[2 * 4 + 0] * in[0] +
		mat[2 * 4 + 1] * in[1] +
		mat[2 * 4 + 2] * in[2] +
		mat[2 * 4 + 3];
	return;
}


__device__ __inline__ int cuda_reg_floor(float a) {
	return a > 0 ? (int)a : (int)(a - 1);
}

template <class FieldTYPE>
__device__ __inline__ void interpolantCubicSpline(FieldTYPE ratio, FieldTYPE *basis) {
	if (ratio < 0.0f) ratio = 0.0f; //reg_rounding error
	FieldTYPE FF = ratio*ratio;
	basis[0] = (FieldTYPE)((ratio * ((2.0f - ratio)*ratio - 1.0f)) / 2.0f);
	basis[1] = (FieldTYPE)((FF * (3.0f*ratio - 5.0) + 2.0f) / 2.0f);
	basis[2] = (FieldTYPE)((ratio * ((4.0f - 3.0f*ratio)*ratio + 1.0f)) / 2.0f);
	basis[3] = (FieldTYPE)((ratio - 1.0f) * FF / 2.0f);
}
__device__ __inline__
void reg_mat44_eye(float *mat) {
	mat[0 * 4 + 0] = 1.f;
	mat[0 * 4 + 1] = mat[0 * 4 + 2] = mat[0 * 4 + 3] = 0.f;
	mat[1 * 4 + 1] = 1.f;
	mat[1 * 4 + 0] = mat[1 * 4 + 2] = mat[1 * 4 + 3] = 0.f;
	mat[2 * 4 + 2] = 1.f;
	mat[2 * 4 + 0] = mat[2 * 4 + 1] = mat[2 * 4 + 3] = 0.f;
	mat[3 * 4 + 3] = 1.f;
	mat[3 * 4 + 0] = mat[3 * 4 + 1] = mat[3 * 4 + 2] = 0.f;
}

template <class DTYPE>
__global__ void reg_dti_resampling_postprocessing(DTYPE *inputImage, DTYPE *warpedImage, int *mask, float *jacMat, int *dtiIndeces, uint3 fi_xyz, uint2 ii_tu) {
	// If we have some valid diffusion tensor indicies, we need to exponentiate the previously logged tensor components
	// we also need to reorient the tensors based on the local transformation Jacobians

	//if (dtiIndeces[0] != -1) {

	//	long warpedIndex = blockIdx.x*blockDim.x + threadIdx.x;
	//	long voxelNumber = fi_xyz.x*fi_xyz.y*fi_xyz.z;

	//	DTYPE *warpVox, *warpedXX, *warpedXY, *warpedXZ, *warpedYY, *warpedYZ, *warpedZZ;
	//	if (warpedImage != NULL) {
	//		warpVox = static_cast<DTYPE *>(warpedImage);
	//		warpedXX = &warpVox[voxelNumber*dtiIndeces[0]];
	//		warpedXY = &warpVox[voxelNumber*dtiIndeces[1]];
	//		warpedYY = &warpVox[voxelNumber*dtiIndeces[2]];
	//		warpedXZ = &warpVox[voxelNumber*dtiIndeces[3]];
	//		warpedYZ = &warpVox[voxelNumber*dtiIndeces[4]];
	//		warpedZZ = &warpVox[voxelNumber*dtiIndeces[5]];
	//	}
	//	for (int u = 0; u < ii_tu.y; ++u) {
	//		// Now, we need to exponentiate the warped intensities back to give us a regular tensor
	//		// let's reorient each tensor based on the rigid component of the local warping
	//		/* As the tensor has 6 unique components that we need to worry about, read them out
	//		for the warped image. */
	//		const unsigned int txu = ii_tu.x*ii_tu.y;

	//		DTYPE *firstWarpVox = static_cast<DTYPE *>(inputImage);
	//		DTYPE *inputIntensityXX = &firstWarpVox[voxelNumber*(dtiIndeces[0] + txu)];
	//		DTYPE *inputIntensityXY = &firstWarpVox[voxelNumber*(dtiIndeces[1] + txu)];
	//		DTYPE *inputIntensityYY = &firstWarpVox[voxelNumber*(dtiIndeces[2] + txu)];
	//		DTYPE *inputIntensityXZ = &firstWarpVox[voxelNumber*(dtiIndeces[3] + txu)];
	//		DTYPE *inputIntensityYZ = &firstWarpVox[voxelNumber*(dtiIndeces[4] + txu)];
	//		DTYPE *inputIntensityZZ = &firstWarpVox[voxelNumber*(dtiIndeces[5] + txu)];

	//		// Step through each voxel in the warped image
	//		double testSum = 0;
	//		float jacobianMatrix[9], R[9];
	//		float inputTensor[16], warpedTensor[16], RotMat[16], RotMatT[16], preMult[16];
	//		int col, row;

	//		if (mask[warpedIndex] > -1) {
	//			reg_mat44_eye(inputTensor);
	//			// Fill the rest of the mat44 with the tensor components
	//			inputTensor[0 * 4 + 0] = static_cast<double>(inputIntensityXX[warpedIndex]);
	//			inputTensor[0 * 4 + 1] = static_cast<double>(inputIntensityXY[warpedIndex]);
	//			inputTensor[1 * 4 + 0] = inputTensor[0 * 4 + 1];
	//			inputTensor[1 * 4 + 1] = static_cast<double>(inputIntensityYY[warpedIndex]);
	//			inputTensor[0 * 4 + 2] = static_cast<double>(inputIntensityXZ[warpedIndex]);
	//			inputTensor[2 * 4 + 0] = inputTensor[0 * 4 + 2];
	//			inputTensor[1 * 4 + 2] = static_cast<double>(inputIntensityYZ[warpedIndex]);
	//			inputTensor[2 * 4 + 1] = inputTensor[1 * 4 + 2];
	//			inputTensor[2 * 4 + 2] = static_cast<double>(inputIntensityZZ[warpedIndex]);
	//			// Exponentiate the warped tensor
	//			if (warpedImage == NULL) {
	//				inputTensor[3 * 4 + 3] = static_cast<double>(0.0);
	//				reg_mat44_expm_cuda(inputTensor);
	//				testSum = 0.;
	//			}
	//			else {
	//				inputTensor[3 * 4 + 3] = 1.0;
	//				reg_mat44_eye(warpedTensor);
	//				warpedTensor[0 * 4 + 0] = static_cast<double>(warpedXX[warpedIndex]);
	//				warpedTensor[0 * 4 + 1] = static_cast<double>(warpedXY[warpedIndex]);
	//				warpedTensor[1 * 4 + 0] = warpedTensor[0 * 4 + 1];
	//				warpedTensor[1 * 4 + 1] = static_cast<double>(warpedYY[warpedIndex]);
	//				warpedTensor[0 * 4 + 2] = static_cast<double>(warpedXZ[warpedIndex]);
	//				warpedTensor[2 * 4 + 0] = warpedTensor[0 * 4 + 2];
	//				warpedTensor[1 * 4 + 2] = static_cast<double>(warpedYZ[warpedIndex]);
	//				warpedTensor[2 * 4 + 1] = warpedTensor[1 * 4 + 2];
	//				warpedTensor[2 * 4 + 2] = static_cast<double>(warpedZZ[warpedIndex]);
	//				reg_mat44_mul_cuda<DTYPE>(warpedTensor, inputTensor, inputTensor);
	//				testSum = static_cast<double>(warpedTensor[0 * 4 + 0] + warpedTensor[0 * 4 + 1] + warpedTensor[0 * 4 + 2] +
	//					warpedTensor[1 * 4 + 0] + warpedTensor[1 * 4 + 1] + warpedTensor[1 * 4 + 2] +
	//					warpedTensor[2 * 4 + 0] + warpedTensor[2 * 4 + 1] + warpedTensor[2 * 4 + 2]);
	//			}

	//			if (testSum == testSum) {
	//				// Find the rotation matrix from the local warp Jacobian
	//				jacobianMatrix = jacMat[warpedIndex];
	//				// Calculate the polar decomposition of the local Jacobian matrix, which
	//				// tells us how to rotate the local tensor information
	//				R = nifti_mat33_polar(jacobianMatrix);
	//				// We need both the rotation matrix, and it's transpose as a mat44
	//				reg_mat44_eye(&RotMat);
	//				reg_mat44_eye(&RotMatT);
	//				for (col = 0; col < 3; col++) {
	//					for (row = 0; row < 3; row++) {
	//						RotMat[col * 4 + row] = static_cast<double>(R[col * 4 + row]);
	//						RotMatT[col * 4 + row] = static_cast<double>(R[row * 4 + col]);
	//					}
	//				}
	//				// As the mat44 multiplication uses pointers, do the multiplications separately
	//				reg_mat44_mul_cuda(RotMatT, inputTensor, preMult);
	//				reg_mat44_mul_cuda(preMult, RotMat, inputTensor);

	//				// Finally, read the tensor back out as a warped image
	//				inputIntensityXX[warpedIndex] = static_cast<DTYPE>(inputTensor[0 * 4 + 0]);
	//				inputIntensityYY[warpedIndex] = static_cast<DTYPE>(inputTensor[1 * 4 + 1]);
	//				inputIntensityZZ[warpedIndex] = static_cast<DTYPE>(inputTensor[2 * 4 + 2]);
	//				inputIntensityXY[warpedIndex] = static_cast<DTYPE>(inputTensor[0 * 4 + 1]);
	//				inputIntensityXZ[warpedIndex] = static_cast<DTYPE>(inputTensor[0 * 4 + 2]);
	//				inputIntensityYZ[warpedIndex] = static_cast<DTYPE>(inputTensor[1 * 4 + 2]);
	//			}
	//			else {
	//				inputIntensityXX[warpedIndex] = 0;
	//				inputIntensityYY[warpedIndex] = 0;
	//				inputIntensityZZ[warpedIndex] = 0;
	//				inputIntensityXY[warpedIndex] = 0;
	//				inputIntensityXZ[warpedIndex] = 0;
	//				inputIntensityYZ[warpedIndex] = 0;
	//			}
	//		}

	//	}
	//}
}

template <class DTYPE>
__global__
void reg_dti_resampling_preprocessing(DTYPE *floatingImage, int *dtiIndeces, uint3 fi_xyz) {
	// If we have some valid diffusion tensor indicies, we need to replace the tensor components
	// by the the log tensor components

	//if (dtiIndeces[0] != -1) {

	//	long floatingIndex;
	//	long floatingVoxelNumber = (long)fi_xyz.x*fi_xyz.y*fi_xyz.z;


	//	/* As the tensor has 6 unique components that we need to worry about, read them out
	//	for the floating image. */
	//	DTYPE *firstVox = static_cast<DTYPE *>(floatingImage);
	//	DTYPE *floatingIntensityXX = &firstVox[floatingVoxelNumber*dtiIndeces[0]];
	//	DTYPE *floatingIntensityXY = &firstVox[floatingVoxelNumber*dtiIndeces[1]];
	//	DTYPE *floatingIntensityYY = &firstVox[floatingVoxelNumber*dtiIndeces[2]];
	//	DTYPE *floatingIntensityXZ = &firstVox[floatingVoxelNumber*dtiIndeces[3]];
	//	DTYPE *floatingIntensityYZ = &firstVox[floatingVoxelNumber*dtiIndeces[4]];
	//	DTYPE *floatingIntensityZZ = &firstVox[floatingVoxelNumber*dtiIndeces[5]];

	//	// We need a mat44 to store the diffusion tensor at each voxel for our calculating. Although the DT is 3x3 really,
	//	// it is convenient to store it as a 4x4 to work with existing code for the matrix logarithm/exponential
	//	float diffTensor[16];

	//	// Should log the tensor up front
	//	// We need to take the logarithm of the tensor for each voxel in the floating intensity image, and replace the warped

	//	long index = blockIdx.x*blockDim.x + threadIdx.x;
	//	// Check that the tensor component is not extremely small or extremely large
	//	if ((floatingIntensityXX[floatingIndex] > 1e-10) && (floatingIntensityXX[floatingIndex] < 1e10)) {
	//		// Fill a mat44 with the tensor components
	//		//reg_mat44_eye(&diffTensor);
	//		diffTensor[0 * 4 + 0] = floatingIntensityXX[floatingIndex];
	//		diffTensor[0 * 4 + 1] = floatingIntensityXY[floatingIndex];
	//		diffTensor[1 * 4 + 0] = diffTensor[0 * 4 + 1];
	//		diffTensor[1 * 4 + 1] = floatingIntensityYY[floatingIndex];
	//		diffTensor[0 * 4 + 2] = floatingIntensityXZ[floatingIndex];
	//		diffTensor[2 * 4 + 0] = diffTensor[0 * 4 + 2];
	//		diffTensor[1 * 4 + 2] = floatingIntensityYZ[floatingIndex];
	//		diffTensor[2 * 4 + 1] = diffTensor[1 * 4 + 2];
	//		diffTensor[2 * 4 + 2] = floatingIntensityZZ[floatingIndex];
	//		// Decompose the mat33 into a rotation and a diagonal matrix of eigen values
	//		// Recompose as a log tensor Rt log(E) R, where E is a diagonal matrix
	//		// containing the eigen values and R is a rotation matrix. This is the same as
	//		// taking the logarithm of the tensor
	//		reg_mat44_logm_cuda(diffTensor);
	//		// Write this out as a new image
	//		floatingIntensityXX[floatingIndex] = static_cast<DTYPE>(diffTensor[0 * 4 + 0]);
	//		floatingIntensityXY[floatingIndex] = static_cast<DTYPE>(diffTensor[0 * 4 + 1]);
	//		floatingIntensityYY[floatingIndex] = static_cast<DTYPE>(diffTensor[1 * 4 + 1]);
	//		floatingIntensityXZ[floatingIndex] = static_cast<DTYPE>(diffTensor[0 * 4 + 2]);
	//		floatingIntensityYZ[floatingIndex] = static_cast<DTYPE>(diffTensor[1 * 4 + 2]);
	//		floatingIntensityZZ[floatingIndex] = static_cast<DTYPE>(diffTensor[2 * 4 + 2]);
	//	}
	//	else  // if junk diffusion data, set the diagonal to log the minimum value
	//	{
	//		floatingIntensityXX[floatingIndex] = static_cast<DTYPE>(-23.02585f);
	//		floatingIntensityYY[floatingIndex] = static_cast<DTYPE>(-23.02585f);
	//		floatingIntensityZZ[floatingIndex] = static_cast<DTYPE>(-23.02585f);
	//		floatingIntensityXY[floatingIndex] = static_cast<DTYPE>(0.0f);
	//		floatingIntensityXZ[floatingIndex] = static_cast<DTYPE>(0.0f);
	//		floatingIntensityYZ[floatingIndex] = static_cast<DTYPE>(0.0f);
	//	}
	//}
}

__global__ void CubicSplineResampleImage3D(float *floatingImage, float *deformationField, float *warpedImage, int *mask, /*mat44*/float* sourceIJKMatrix, long2 voxelNumber, uint3 fi_xyz, uint2 wi_tu, float paddingValue) {
	//long resultVoxelNumber = (long)warpedImage->nx*warpedImage->ny*warpedImage->nz;vn.x
	//long sourceVoxelNumber = (long)floatingImage->nx*floatingImage->ny*floatingImage->nz;vn.y

	float *sourceIntensityPtr = (floatingImage);
	float *resultIntensityPtr = (warpedImage);
	float *deformationFieldPtrX = (deformationField);
	float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber.x];
	float *deformationFieldPtrZ = &deformationFieldPtrY[voxelNumber.x];

	int *maskPtr = &mask[0];
	long index = blockIdx.x*blockDim.x + threadIdx.x;
	while (index < voxelNumber.x) {

		// Iteration over the different volume along the 4th axis
		for (unsigned int t = 0; t < wi_tu.x*wi_tu.y; t++) {


			float *resultIntensity = &resultIntensityPtr[t*voxelNumber.x];
			float *sourceIntensity = &sourceIntensityPtr[t*voxelNumber.y];

			float xBasis[4], yBasis[4], zBasis[4], relative;
			int a, b, c, Y, Z, previous[3];

			float *zPointer, *yzPointer, *xyzPointer;
			float xTempNewValue, yTempNewValue, intensity, world[3], position[3];



			intensity = (0.0f);

			if ((maskPtr[index]) > -1) {
				world[0] = deformationFieldPtrX[index];
				world[1] = deformationFieldPtrY[index];
				world[2] = deformationFieldPtrZ[index];

				/* real -> voxel; source space */
				reg_mat44_mul_cuda(sourceIJKMatrix, world, position);

				previous[0] = (cuda_reg_floor(position[0]));
				previous[1] = (cuda_reg_floor(position[1]));
				previous[2] = (cuda_reg_floor(position[2]));

				// basis values along the x axis
				relative = position[0] - previous[0];
				relative = relative > 0 ? relative : 0;
				interpolantCubicSpline<float>(relative, xBasis);
				// basis values along the y axis
				relative = position[1] - previous[1];
				relative = relative > 0 ? relative : 0;
				interpolantCubicSpline<float>(relative, yBasis);
				// basis values along the z axis
				relative = position[2] - previous[2];
				relative = relative > 0 ? relative : 0;
				interpolantCubicSpline<float>(relative, zBasis);

				--previous[0];
				--previous[1];
				--previous[2];

				for (c = 0; c < 4; c++) {
					Z = previous[2] + c;
					zPointer = &sourceIntensity[Z*fi_xyz.x*fi_xyz.y];
					yTempNewValue = 0.0;
					for (b = 0; b < 4; b++) {
						Y = previous[1] + b;
						yzPointer = &zPointer[Y*fi_xyz.x];
						xyzPointer = &yzPointer[previous[0]];
						xTempNewValue = 0.0;
						for (a = 0; a < 4; a++) {
							if (-1 < (previous[0] + a) && (previous[0] + a) < fi_xyz.x &&
								-1 < Z && Z < fi_xyz.z &&
								-1 < Y && Y < fi_xyz.y) {
								xTempNewValue += *xyzPointer * xBasis[a];
							}
							else {
								// paddingValue
								xTempNewValue += paddingValue * xBasis[a];
							}
							xyzPointer++;
						}
						yTempNewValue += xTempNewValue * yBasis[b];
					}
					intensity += yTempNewValue * zBasis[c];
				}
			}

			resultIntensity[index] = intensity;
		}
		index += blockDim.x*gridDim.x;
	}
}

/* *************************************************************** */
__global__ void NearestNeighborResampleImage(float *floatingImage, float *deformationField, float *warpedImage, int *mask, /*mat44*/float* sourceIJKMatrix, long2 voxelNumber, uint3 fi_xyz, uint2 wi_tu, float paddingValue) {

	// The resampling scheme is applied along each time

	float *sourceIntensityPtr = (floatingImage);
	float *resultIntensityPtr = (warpedImage);
	float *deformationFieldPtrX = (deformationField);
	float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber.x];
	float *deformationFieldPtrZ = &deformationFieldPtrY[voxelNumber.x];

	int *maskPtr = &mask[0];


	long index = blockIdx.x*blockDim.x + threadIdx.x;
	while (index < voxelNumber.x) {
		for (int t = 0; t<wi_tu.x*wi_tu.x; t++) {

			float *resultIntensity = &resultIntensityPtr[t*voxelNumber.x];
			float *sourceIntensity = &sourceIntensityPtr[t*voxelNumber.y];

			float intensity;
			float world[3];
			float position[3];
			int previous[3];

			if (maskPtr[index]>-1) {
				world[0] = (float)deformationFieldPtrX[index];
				world[1] = (float)deformationFieldPtrY[index];
				world[2] = (float)deformationFieldPtrZ[index];

				/* real -> voxel; source space */
				reg_mat44_mul_cuda(sourceIJKMatrix, world, position);

				previous[0] = (int)reg_round(position[0]);
				previous[1] = (int)reg_round(position[1]);
				previous[2] = (int)reg_round(position[2]);

				if (-1 < previous[2] && previous[2] < fi_xyz.z &&
					-1 < previous[1] && previous[1] < fi_xyz.y &&
					-1 < previous[0] && previous[0] < fi_xyz.x) {
					intensity = sourceIntensity[(previous[2] * fi_xyz.y + previous[1]) * fi_xyz.x + previous[0]];
					resultIntensity[index] = intensity;
				}
				else resultIntensity[index] = paddingValue;
			}
			else resultIntensity[index] = paddingValue;


		}
		index += blockDim.x*gridDim.x;
	}

}

__global__ void TrilinearResampleImage(float *floatingImage, float *deformationField, float *warpedImage, int *mask, /*mat44*/float* sourceIJKMatrix, long2 voxelNumber, uint3 fi_xyz, uint2 wi_tu, float paddingValue) {

	//if( threadIdx.x == 0 ) printf("block: %d \n", blockIdx.x);

	//targetVoxelNumber voxelNumber.x
	// sourceVoxelNumber voxelNumber.y

	//intensity images
	float *sourceIntensityPtr = (floatingImage);//best to be a texture
	float *resultIntensityPtr = (warpedImage);

	//deformation field image
	float *deformationFieldPtrX = (deformationField);
	float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber.x];
	float *deformationFieldPtrZ = &deformationFieldPtrY[voxelNumber.x];

	int *maskPtr = &mask[0];

	// The resampling scheme is applied along each time

	long index = blockIdx.x*blockDim.x + threadIdx.x;
	while (index < voxelNumber.x) {
		for (unsigned int t = 0; t<wi_tu.x*wi_tu.y; t++) {


			float *resultIntensity = &resultIntensityPtr[t*voxelNumber.x];
			float *sourceIntensity = &sourceIntensityPtr[t*voxelNumber.y];

			float xBasis[2], yBasis[2], zBasis[2], relative;
			int a, b, c, X, Y, Z, previous[3];

			float *zPointer, *xyzPointer;
			float xTempNewValue, yTempNewValue, intensity, world[3], position[3];

			//for( index = 0; index<targetVoxelNumber; index++ ) {

			intensity = paddingValue;

			if (maskPtr[index]>-1) {

				intensity = 0;

				world[0] = deformationFieldPtrX[index];
				world[1] = deformationFieldPtrY[index];
				world[2] = deformationFieldPtrZ[index];

				/* real -> voxel; source space */
				reg_mat44_mul_cuda<float>(sourceIJKMatrix, world, position);

				previous[0] = cuda_reg_floor(position[0]);
				previous[1] = cuda_reg_floor(position[1]);
				previous[2] = cuda_reg_floor(position[2]);

				// basis values along the x axis
				relative = position[0] - previous[0];
				xBasis[0] = (1.0 - relative);
				xBasis[1] = relative;
				// basis values along the y axis
				relative = position[1] - previous[1];
				yBasis[0] = (1.0 - relative);
				yBasis[1] = relative;
				// basis values along the z axis
				relative = position[2] - previous[2];
				zBasis[0] = (1.0 - relative);
				zBasis[1] = relative;

				// For efficiency reason two interpolation are here, with and without using a padding value
				if (paddingValue==paddingValue) {
					// Interpolation using the padding value
					for (c = 0; c<2; c++) {
						Z = previous[2] + c;
						if (Z>-1 && Z < fi_xyz.z) {
							zPointer = &sourceIntensity[Z*fi_xyz.x*fi_xyz.y];
							yTempNewValue = 0.0;
							for (b = 0; b<2; b++) {
								Y = previous[1] + b;
								if (Y>-1 && Y < fi_xyz.y) {
									xyzPointer = &zPointer[Y*fi_xyz.x + previous[0]];
									xTempNewValue = 0.0;
									for (a = 0; a<2; a++) {
										X = previous[0] + a;
										if (X>-1 && X < fi_xyz.x) {
											xTempNewValue += *xyzPointer * xBasis[a];
										} // X
										else xTempNewValue += paddingValue * xBasis[a];
										xyzPointer++;
									} // a
									yTempNewValue += xTempNewValue * yBasis[b];
								} // Y
								else yTempNewValue += paddingValue * yBasis[b];
							} // b
							intensity += yTempNewValue * zBasis[c];
						} // Z
						else intensity += paddingValue * zBasis[c];
					} // c
				} // padding value is defined
				else if (previous[0] >= 0.f && previous[0] < (fi_xyz.x - 1) &&
					previous[1] >= 0.f && previous[1] < (fi_xyz.y - 1) &&
					previous[2] >= 0.f && previous[2] < (fi_xyz.z - 1)) {
					for (c = 0; c < 2; c++) {
						Z = previous[2] + c;
						zPointer = &sourceIntensity[Z*fi_xyz.x*fi_xyz.y];
						yTempNewValue = 0.0;
						for (b = 0; b < 2; b++) {
							Y = previous[1] + b;
							xyzPointer = &zPointer[Y*fi_xyz.x + previous[0]];
							xTempNewValue = 0.0;
							for (a = 0; a < 2; a++) {
								X = previous[0] + a;
								xTempNewValue += *xyzPointer * xBasis[a];
								xyzPointer++;
							} // a
							yTempNewValue += xTempNewValue * yBasis[b];
						} // b
						intensity += yTempNewValue * zBasis[c];
					} // c
				} // padding value is not defined
				// The voxel is outside of the source space and thus set to NaN here
				else intensity = paddingValue;
			} // voxel is in the mask

			resultIntensity[index] = intensity;

			//}
		}
		index += blockDim.x*gridDim.x;
	}

}


__device__ __inline__ void getPosition(float* position, float* matrix, float* voxel, const unsigned int idx) {
	position[idx] =
		matrix[idx * 4 + 0] * voxel[0] +
		matrix[idx * 4 + 1] * voxel[1] +
		matrix[idx * 4 + 2] * voxel[2] +
		matrix[idx * 4 + 3];
}

__global__ void affineKernel(float* transformationMatrix, float* defField, int* mask, const uint3 params, const unsigned long voxelNumber, const bool composition) {

	float *deformationFieldPtrX = defField;
	float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber];
	float *deformationFieldPtrZ = &deformationFieldPtrY[voxelNumber];

	float voxel[3], position[3];


	const unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned long index = x + y*params.x + z * params.x * params.y;
	if (z < params.z && y < params.y && x < params.x &&  mask[index] >= 0) {

		voxel[0] = composition ? deformationFieldPtrX[index] : x;
		voxel[1] = composition ? deformationFieldPtrY[index] : y;
		voxel[2] = composition ? deformationFieldPtrZ[index] : z;

		getPosition(position, transformationMatrix, voxel, 0);
		getPosition(position, transformationMatrix, voxel, 1);
		getPosition(position, transformationMatrix, voxel, 2);

		/* the deformation field (real coordinates) is stored */
		deformationFieldPtrX[index] = position[0];
		deformationFieldPtrY[index] = position[1];
		deformationFieldPtrZ[index] = position[2];

	}
}

template<class DTYPE>
__global__ void convolutionKernel(nifti_image *image, float*densityPtr, bool* nanImagePtr, float *size, int kernelType, int *mask, bool *timePoint, bool *axis) {
	if (threadIdx.x == 0) {
		//printf("hi from %d-%d \n", blockIdx.x, threadIdx.x);
		const unsigned long voxelNumber = image->dim[1] * image->dim[2] * image->dim[3];
		DTYPE *imagePtr = static_cast<DTYPE *>(image->data);
		int imageDim[3] = { image->dim[1], image->dim[2], image->dim[3] };


		// Loop over the dimension higher than 3
		for (int t = 0; t < image->dim[4] * image->dim[5]; t++) {
			if (timePoint[t]) {
				DTYPE *intensityPtr = &imagePtr[t * voxelNumber];

				for (unsigned long index = 0; index < voxelNumber; index++) {
					densityPtr[index] = (intensityPtr[index] == intensityPtr[index]) ? 1 : 0;
					densityPtr[index] *= (mask[index] >= 0) ? 1 : 0;
					nanImagePtr[index] = static_cast<bool>(densityPtr[index]);
					if (nanImagePtr[index] == 0)
						intensityPtr[index] = static_cast<DTYPE>(0);
				}
				// Loop over the x, y and z dimensions
				for (int n = 0; n < 3; n++) {
					if (axis[n] && image->dim[n] > 1) {
						double temp;
						if (size[t]>0) temp = size[t] / image->pixdim[n + 1]; // mm to voxel
						else temp = fabs(size[t]); // voxel based if negative value
						int radius;
						// Define the kernel size
						if (kernelType == 2) {
							// Mean filtering
							radius = static_cast<int>(temp);
						}
						else if (kernelType == 1) {
							// Cubic Spline kernel
							radius = static_cast<int>(temp*2.0f);
						}
						else {
							// Gaussian kernel
							radius = static_cast<int>(temp*3.0f);
						}
						if (radius > 0) {
							// Allocate the kernel
							float kernel[2048];
							double kernelSum = 0;
							// Fill the kernel
							if (kernelType == 1) {
								// Compute the Cubic Spline kernel
								for (int i = -radius; i <= radius; i++) {
									// temp contains the kernel node spacing
									double relative = (double)(fabs((double)(double)i / (double)temp));
									if (relative < 1.0) kernel[i + radius] = (float)(2.0 / 3.0 - relative*relative + 0.5*relative*relative*relative);
									else if (relative < 2.0) kernel[i + radius] = (float)(-(relative - 2.0)*(relative - 2.0)*(relative - 2.0) / 6.0);
									else kernel[i + radius] = 0;
									kernelSum += kernel[i + radius];
								}
							}
							// No kernel is required for the mean filtering
							else if (kernelType != 2) {
								// Compute the Gaussian kernel
								for (int i = -radius; i <= radius; i++) {
									// 2.506... = sqrt(2*pi)
									// temp contains the sigma in voxel
									kernel[radius + i] = static_cast<float>(exp(-(double)(i*i) / (2.0*reg_pow2(temp))) /
										(temp*2.506628274631));
									kernelSum += kernel[radius + i];
								}
							}
							// No need for kernel normalisation as this is handle by the density function
							int planeNumber, planeIndex, lineOffset;
							int lineIndex, shiftPre, shiftPst, k;
							switch (n) {
							case 0:
								planeNumber = imageDim[1] * imageDim[2];
								lineOffset = 1;
								break;
							case 1:
								planeNumber = imageDim[0] * imageDim[2];
								lineOffset = imageDim[0];
								break;
							case 2:
								planeNumber = imageDim[0] * imageDim[1];
								lineOffset = planeNumber;
								break;
							}

							size_t realIndex;
							float *kernelPtr, kernelValue;
							double densitySum, intensitySum;
							DTYPE *currentIntensityPtr = NULL;
							float *currentDensityPtr = NULL;
							DTYPE bufferIntensity[2048];;
							float bufferDensity[2048];
							DTYPE bufferIntensitycur = 0;
							float bufferDensitycur = 0;

							// Loop over the different voxel
							for (planeIndex = 0; planeIndex < planeNumber; ++planeIndex) {

								switch (n) {
								case 0:
									realIndex = planeIndex * imageDim[0];
									break;
								case 1:
									realIndex = (planeIndex / imageDim[0]) *
										imageDim[0] * imageDim[1] +
										planeIndex%imageDim[0];
									break;
								case 2:
									realIndex = planeIndex;
									break;
								default:
									realIndex = 0;
								}
								// Fetch the current line into a stack buffer
								currentIntensityPtr = &intensityPtr[realIndex];
								currentDensityPtr = &densityPtr[realIndex];
								for (lineIndex = 0; lineIndex < imageDim[n]; ++lineIndex) {
									bufferIntensity[lineIndex] = *currentIntensityPtr;
									bufferDensity[lineIndex] = *currentDensityPtr;
									currentIntensityPtr += lineOffset;
									currentDensityPtr += lineOffset;
								}
								if (kernelSum > 0) {
									// Perform the kernel convolution along 1 line
									for (lineIndex = 0; lineIndex < imageDim[n]; ++lineIndex) {
										// Define the kernel boundaries
										shiftPre = lineIndex - radius;
										shiftPst = lineIndex + radius + 1;
										if (shiftPre < 0) {
											kernelPtr = &kernel[-shiftPre];
											shiftPre = 0;
										}
										else kernelPtr = &kernel[0];
										if (shiftPst > imageDim[n]) shiftPst = imageDim[n];
										// Set the current values to zero
										intensitySum = 0;
										densitySum = 0;
										// Increment the current value by performing the weighted sum
										for (k = shiftPre; k < shiftPst; ++k) {
											kernelValue = *kernelPtr++;
											intensitySum += kernelValue * bufferIntensity[k];
											densitySum += kernelValue * bufferDensity[k];
										}
										// Store the computed value inplace
										intensityPtr[realIndex] = static_cast<DTYPE>(intensitySum);
										densityPtr[realIndex] = static_cast<float>(densitySum);
										realIndex += lineOffset;
									} // line convolution
								} // kernel type
								else {
									for (lineIndex = 1; lineIndex < imageDim[n]; ++lineIndex) {
										bufferIntensity[lineIndex] += bufferIntensity[lineIndex - 1];
										bufferDensity[lineIndex] += bufferDensity[lineIndex - 1];
									}
									shiftPre = -radius - 1;
									shiftPst = radius;
									for (lineIndex = 0; lineIndex < imageDim[n]; ++lineIndex, ++shiftPre, ++shiftPst) {
										if (shiftPre > -1) {
											if (shiftPst < imageDim[n]) {
												bufferIntensitycur = (DTYPE)(bufferIntensity[shiftPre] - bufferIntensity[shiftPst]);
												bufferDensitycur = (DTYPE)(bufferDensity[shiftPre] - bufferDensity[shiftPst]);
											}
											else {
												bufferIntensitycur = (DTYPE)(bufferIntensity[shiftPre] - bufferIntensity[imageDim[n] - 1]);
												bufferDensitycur = (DTYPE)(bufferDensity[shiftPre] - bufferDensity[imageDim[n] - 1]);
											}
										}
										else {
											if (shiftPst < imageDim[n]) {
												bufferIntensitycur = (DTYPE)(-bufferIntensity[shiftPst]);
												bufferDensitycur = (DTYPE)(-bufferDensity[shiftPst]);
											}
											else {
												bufferIntensitycur = (DTYPE)(0);
												bufferDensitycur = (DTYPE)(0);
											}
										}
										intensityPtr[realIndex] = bufferIntensitycur;
										densityPtr[realIndex] = bufferDensitycur;

										realIndex += lineOffset;
									} // line convolution of mean filter
								} // No kernel computation
							} // pixel in starting plane
						} // radius > 0
					} // active axis
				} // axes
				// Normalise per timepoint
				for (unsigned long index = 0; index < voxelNumber; ++index) {
					if (nanImagePtr[index] != 0)
						intensityPtr[index] = static_cast<DTYPE>((float)intensityPtr[index] / densityPtr[index]);
					else intensityPtr[index] = 0;
				}
			} // check if the time point is active
		} // loop over the time points
	}
}

void launch(nifti_image *image, float *sigma, int kernelType, int *mask, bool *timePoint, bool *axis) {
	bool *axisToSmooth = new bool[3];
	bool *activeTimePoint = new bool[image->nt*image->nu];
	unsigned long voxelNumber = (long)image->nx*image->ny*image->nz;

	bool *nanImagePtr;
	float *densityPtr;
	float *sigma_d;
	int *mask_d;
	bool* timePoint_d;
	bool* axis_d;


	int dim[3] = { image->nx, image->ny, image->nz };
	std::cout << image->nx << ": " << image->ny << ": " << image->nz << std::endl;
	nifti_image* image_d;


	if (image->nx > 2048 || image->ny > 2048 || image->nz > 2048) {
		reg_print_fct_error("reg_tools_kernelConvolution_core");
		reg_print_msg_error("This function does not support images with dimension > 2048");
		reg_exit(1);
	}

	if (image->nt <= 0) image->nt = image->dim[4] = 1;
	if (image->nu <= 0) image->nu = image->dim[5] = 1;




	/*densityPtr[4] = 8.8f;
	std::cout << "test float: " << densityPtr[4] << std::endl;*/


	if (axis == NULL) {
		// All axis are smoothed by default
		for (int i = 0; i < 3; i++) axisToSmooth[i] = true;
	}
	else for (int i = 0; i < 3; i++) axisToSmooth[i] = axis[i];

	if (timePoint == NULL) {
		// All time points are considered as active
		for (int i = 0; i < image->nt*image->nu; i++) activeTimePoint[i] = true;
	}
	else for (int i = 0; i < image->nt*image->nu; i++) activeTimePoint[i] = timePoint[i];

	int *currentMask = NULL;
	if (mask == NULL) {
		currentMask = (int *)calloc(image->nx*image->ny*image->nz, sizeof(int));
	}
	else currentMask = mask;

	/*cudaCommon_allocateNiftiToDevice<float>(&image_d, dim);
	cudaCommon_transferNiftiToNiftiOnDevice1<float>(&image_d, image);*/

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(sigma_d), image->dim[4] * image->dim[5] * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(sigma_d, sigma, image->dim[4] * image->dim[5] * sizeof(float), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(mask_d), voxelNumber * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(mask_d, currentMask, voxelNumber * sizeof(int), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(timePoint_d), image->dim[4] * image->dim[5] * sizeof(bool)));
	NR_CUDA_SAFE_CALL(hipMemcpy(timePoint_d, timePoint, image->dim[4] * image->dim[5] * sizeof(bool), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(axis_d), 3 * sizeof(bool)));
	NR_CUDA_SAFE_CALL(hipMemcpy(axis_d, axis, 3 * sizeof(bool), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc(&nanImagePtr, voxelNumber*sizeof(bool)));
	NR_CUDA_SAFE_CALL(hipMalloc(&densityPtr, voxelNumber*sizeof(float)));

	switch (image->datatype) {
	case NIFTI_TYPE_UINT8:
		//convolutionKernel<unsigned char> <<<1, 1 >>>( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_INT8:
		//convolutionKernel <char> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_UINT16:
		//convolutionKernel <unsigned short> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_INT16:
		//convolutionKernel <short> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_UINT32:
		//convolutionKernel<unsigned int> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_INT32:
		//convolutionKernel <int> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	case NIFTI_TYPE_FLOAT32:
		std::cout << "called instead of kernel!" << std::endl;
		convolutionKernel <float> << <1, 1 >> >(image_d, densityPtr, nanImagePtr, sigma_d, kernelType, mask_d, timePoint_d, axis_d);
		//NR_CUDA_CHECK_KERNEL(1, 1)
		break;
	case NIFTI_TYPE_FLOAT64:
		//convolutionKernel <double> << <1, 1 >> >( image, densityPtr, nanImagePtr, sigma, kernelType, currentMask, activeTimePoint, axisToSmooth );
		break;
	default:
		fprintf(stderr, "[NiftyReg ERROR] reg_gaussianSmoothing\tThe image data type is not supported\n");
		reg_exit(1);
	}

	if (mask == NULL) free(currentMask);
	delete[]axisToSmooth;
	delete[]activeTimePoint;
	hipFree(nanImagePtr);
	hipFree(densityPtr);
}




nifti_params_t getParams(nifti_image image) {
	nifti_params_t params = {
		image.ndim,                    /*!< last dimension greater than 1 (1..7) */
		image.nx,                      /*!< dimensions of grid array             */
		image.ny,                      /*!< dimensions of grid array             */
		image.nz,                      /*!< dimensions of grid array             */
		image.nt,                      /*!< dimensions of grid array             */
		image.nu,                      /*!< dimensions of grid array             */
		image.nv,                      /*!< dimensions of grid array             */
		image.nw,                      /*!< dimensions of grid array             */
		image.nvox,					   /*!< number of voxels = nx*ny*nz*...*nw   */
		image.nbyper,                  /*!< bytes per voxel, matches datatype    */
		image.datatype,                /*!< type of data in voxels: DT_* code    */

		image.dx,					/*!< grid spacings      */
		image.dy,                   /*!< grid spacings      */
		image.dz,                   /*!< grid spacings      */
		image.dt,                   /*!< grid spacings      */
		image.du,                   /*!< grid spacings      */
		image.dv,                   /*!< grid spacings      */
		image.dw,                    /*!< grid spacings      */
		image.nx*image.ny*image.nz   //xyz image size
	};

	return params;
}
void launchAffine(mat44 *affineTransformation, nifti_image *deformationField, bool compose, int *mask) {

	const unsigned int xThreads = 8;
	const unsigned int yThreads = 8;
	const unsigned int zThreads = 8;

	const unsigned int xBlocks = ((deformationField->nx % xThreads) == 0) ? (deformationField->nx / xThreads) : (deformationField->nx / xThreads) + 1;
	const unsigned int yBlocks = ((deformationField->ny % yThreads) == 0) ? (deformationField->ny / yThreads) : (deformationField->ny / yThreads) + 1;
	const unsigned int zBlocks = ((deformationField->nz % zThreads) == 0) ? (deformationField->nz / zThreads) : (deformationField->nz / zThreads) + 1;


	dim3 G1_b(xBlocks, yBlocks, zBlocks);
	dim3 B1_b(xThreads, yThreads, zThreads);


	int *tempMask = mask;
	if (mask == NULL) {
		tempMask = (int *)calloc(deformationField->nx*
			deformationField->ny*
			deformationField->nz,
			sizeof(int));
	}

	const mat44 *targetMatrix = (deformationField->sform_code > 0) ? &(deformationField->sto_xyz) : &(deformationField->qto_xyz);
	mat44 transformationMatrix = (compose == true) ? *affineTransformation : reg_mat44_mul(affineTransformation, targetMatrix);

	float* trans = (float *)malloc(16 * sizeof(float));
	mat44ToCptr(transformationMatrix, trans);

	nifti_params params_d = getParams(*deformationField);
	float *trans_d, *def_d;
	int* mask_d;



	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&trans_d), 16 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(trans_d, trans, 16 * sizeof(float), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&def_d), params_d.nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(def_d, deformationField->data, params_d.nvox * sizeof(float), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&mask_d), params_d.nxyz * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(mask_d, tempMask, params_d.nxyz * sizeof(int), hipMemcpyHostToDevice));



	uint3 pms_d = make_uint3(params_d.nx, params_d.ny, params_d.nz);
	affineKernel << <G1_b, B1_b >> >(trans_d, def_d, mask_d, pms_d, params_d.nxyz, compose);
	NR_CUDA_CHECK_KERNEL(G1_b, B1_b)

		NR_CUDA_SAFE_CALL(hipMemcpy(deformationField->data, def_d, params_d.nvox * sizeof(float), hipMemcpyDeviceToHost));

	if (mask == NULL)
		free(tempMask);

	hipFree(trans_d);
	hipFree(def_d);
	hipFree(mask_d);

}
void launchAffine2(mat44 *affineTransformation, nifti_image *deformationField, float** def_d, int** mask_d, bool compose) {

	const unsigned int xThreads = 8;
	const unsigned int yThreads = 8;
	const unsigned int zThreads = 8;

	const unsigned int xBlocks = ((deformationField->nx % xThreads) == 0) ? (deformationField->nx / xThreads) : (deformationField->nx / xThreads) + 1;
	const unsigned int yBlocks = ((deformationField->ny % yThreads) == 0) ? (deformationField->ny / yThreads) : (deformationField->ny / yThreads) + 1;
	const unsigned int zBlocks = ((deformationField->nz % zThreads) == 0) ? (deformationField->nz / zThreads) : (deformationField->nz / zThreads) + 1;


	dim3 G1_b(xBlocks, yBlocks, zBlocks);
	dim3 B1_b(xThreads, yThreads, zThreads);



	const mat44 *targetMatrix = (deformationField->sform_code > 0) ? &(deformationField->sto_xyz) : &(deformationField->qto_xyz);
	mat44 transformationMatrix = (compose == true) ? *affineTransformation : reg_mat44_mul(affineTransformation, targetMatrix);

	float* trans = (float *)malloc(16 * sizeof(float));
	mat44ToCptr(transformationMatrix, trans);

	nifti_params params_d = getParams(*deformationField);
	float *trans_d;

	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&trans_d), 16 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(trans_d, trans, 16 * sizeof(float), hipMemcpyHostToDevice));

	uint3 pms_d = make_uint3(params_d.nx, params_d.ny, params_d.nz);
	affineKernel << <G1_b, B1_b >> >(trans_d, *def_d, *mask_d, pms_d, params_d.nxyz, compose);
	//NR_CUDA_CHECK_KERNEL(G1_b, B1_b)
	NR_CUDA_SAFE_CALL(hipDeviceSynchronize());
	hipFree(trans_d);
	free(trans);

}
void launchOptimize(_reg_blockMatchingParam *params, mat44 *transformation_matrix, bool affine) {
	float in[3];
	float out[3];
	for (size_t i = 0; i<static_cast<size_t>(params->activeBlockNumber); ++i)
	{
		size_t index = 3 * i;
		in[0] = params->resultPosition[index];
		in[1] = params->resultPosition[index + 1];
		in[2] = params->resultPosition[index + 2];
		reg_mat44_mul(transformation_matrix, in, out);
		params->resultPosition[index++] = out[0];
		params->resultPosition[index++] = out[1];
		params->resultPosition[index] = out[2];
	}
	if (affine)
		launchOptimizeAffine(params, transformation_matrix, true);
	else launchOptimizeRigid(params, transformation_matrix, false);
}

void launchResample(nifti_image *floatingImage, nifti_image *warpedImage, nifti_image *deformationField, int *mask, int interp, float paddingValue, bool *dti_timepoint, mat33 * jacMat) {

	if (floatingImage->datatype != warpedImage->datatype) {
		printf("[NiftyReg ERROR] reg_resampleImage\tSource and result image should have the same data type\n");
		printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
		reg_exit(1);
	}

	if (floatingImage->nt != warpedImage->nt) {
		printf("[NiftyReg ERROR] reg_resampleImage\tThe source and result images have different dimension along the time axis\n");
		printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
		reg_exit(1);
	}

	// Define the DTI indices if required
	int dtiIndeces[6];
	for (int i = 0; i < 6; ++i) dtiIndeces[i] = -1;
	if (dti_timepoint != NULL) {

		if (jacMat == NULL) {
			printf("[NiftyReg ERROR] reg_resampleImage\tDTI resampling\n");
			printf("[NiftyReg ERROR] reg_resampleImage\tNo Jacobian matrix array has been provided\n");
			reg_exit(1);
		}
		int j = 0;
		for (int i = 0; i < floatingImage->nt; ++i) {
			if (dti_timepoint[i] == true)
				dtiIndeces[j++] = i;
		}
		if ((floatingImage->nz>1 && j != 6) && (floatingImage->nz == 1 && j != 3)) {
			printf("[NiftyReg ERROR] reg_resampleImage\tUnexpected number of DTI components\n");
			printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
			reg_exit(1);
		}
	}

	// a mask array is created if no mask is specified
	bool MrPropreRules = false;
	if (mask == NULL) {
		// voxels in the backgreg_round are set to -1 so 0 will do the job here
		mask = (int *)calloc(warpedImage->nx*warpedImage->ny*warpedImage->nz, sizeof(int));
		MrPropreRules = true;
	}


	runKernel(floatingImage, warpedImage, deformationField, mask, interp, paddingValue, dtiIndeces, jacMat);

	if (MrPropreRules == true) {
		free(mask);
		mask = NULL;
	}
}
void launchResample2(nifti_image *floatingImage, nifti_image *warpedImage, int *mask, int interp, float paddingValue, bool *dti_timepoint, mat33 * jacMat, float** floatingImage_d,  float** warpedImage_d, float** deformationFieldImage_d, int** mask_d) {

	if (floatingImage->datatype != warpedImage->datatype) {
		printf("[NiftyReg ERROR] reg_resampleImage\tSource and result image should have the same data type\n");
		printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
		reg_exit(1);
	}

	if (floatingImage->nt != warpedImage->nt) {
		printf("[NiftyReg ERROR] reg_resampleImage\tThe source and result images have different dimension along the time axis\n");
		printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
		reg_exit(1);
	}

	// Define the DTI indices if required
	int dtiIndeces[6];
	for (int i = 0; i < 6; ++i) dtiIndeces[i] = -1;
	if (dti_timepoint != NULL) {

		if (jacMat == NULL) {
			printf("[NiftyReg ERROR] reg_resampleImage\tDTI resampling\n");
			printf("[NiftyReg ERROR] reg_resampleImage\tNo Jacobian matrix array has been provided\n");
			reg_exit(1);
		}
		int j = 0;
		for (int i = 0; i < floatingImage->nt; ++i) {
			if (dti_timepoint[i] == true)
				dtiIndeces[j++] = i;
		}
		if ((floatingImage->nz>1 && j != 6) && (floatingImage->nz == 1 && j != 3)) {
			printf("[NiftyReg ERROR] reg_resampleImage\tUnexpected number of DTI components\n");
			printf("[NiftyReg ERROR] reg_resampleImage\tNothing has been done\n");
			reg_exit(1);
		}
	}

	// a mask array is created if no mask is specified
	bool MrPropreRules = false;
	if (mask == NULL) {
		// voxels in the backgreg_round are set to -1 so 0 will do the job here
		mask = (int *)calloc(warpedImage->nx*warpedImage->ny*warpedImage->nz, sizeof(int));
		MrPropreRules = true;
	}

	//printf("kernel2run");
	runKernel2(floatingImage, warpedImage, mask, interp, paddingValue, dtiIndeces, jacMat,  floatingImage_d, warpedImage_d, deformationFieldImage_d,  mask_d);

	if (MrPropreRules == true) {
		free(mask);
		mask = NULL;
	}
}

void initTextures() {
	hipArray **floatingImageArray_d;
	//cudaCommon_transferNiftiToArrayOnDevice1(floatingImageArray_d, floatingImage)
	////Bind floating image array to a 3D texture
	//floatingTexture.normalized = false;
	//floatingTexture.filterMode = hipFilterModeLinear;
	//floatingTexture.addressMode[0] = hipAddressModeWrap;
	//floatingTexture.addressMode[1] = hipAddressModeWrap;
	//floatingTexture.addressMode[2] = hipAddressModeWrap;

	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	//NR_CUDA_SAFE_CALL(hipBindTextureToArray(floatingTexture, *floatingImageArray_d, channelDesc))
}

void runKernel(nifti_image *floatingImage, nifti_image *warpedImage, nifti_image *deformationFieldImage, int *mask, int interp, float paddingValue, int *dtiIndeces, mat33 * jacMat) {


	long targetVoxelNumber = (long)warpedImage->nx*warpedImage->ny*warpedImage->nz;
	hipDeviceProp_t  prop;
	hipGetDeviceProperties(&prop, 0);
	unsigned int maxThreads = prop.maxThreadsDim[0];
	unsigned int maxBlocks = prop.maxThreadsDim[0];
	unsigned int blocks = (targetVoxelNumber % maxThreads) ? (targetVoxelNumber / maxThreads) + 1 : targetVoxelNumber / maxThreads;
	blocks = min1(blocks, maxBlocks);



	dim3 mygrid(blocks, 1, 1);
	dim3 myblocks(maxThreads, 1, 1);
	printf("maxBlocks b: %d | maxThreads: %d\n", maxBlocks, maxThreads);
	printf("blocks: %d | threads: %d\n", blocks, maxThreads);

	// The floating image data is copied in case one deal with DTI
	void *originalFloatingData = NULL;
	originalFloatingData = (void *)malloc(floatingImage->nvox*sizeof(float));
	memcpy(originalFloatingData, floatingImage->data, floatingImage->nvox*sizeof(float));


	int numMats = 0;
	mat44 *sourceIJKMatrix;
	float *sourceIJKMatrix_h = (float*)malloc(16 * sizeof(float));
	float* jacMat_h = (float*)malloc(9 * numMats*sizeof(float));

	if (floatingImage->sform_code > 0)
		sourceIJKMatrix = &(floatingImage->sto_ijk);
	else sourceIJKMatrix = &(floatingImage->qto_ijk);

	float *floatingImage_d, *deformationFieldImage_d, *warpedImage_d, paddingValue_d;
	float* sourceIJKMatrix_d, *jacMat_d;
	int* mask_d, *dtiIndeces_d;
	long2 voxelNumber = make_long2(warpedImage->nx*warpedImage->ny*warpedImage->nz, floatingImage->nx*floatingImage->ny*floatingImage->nz);
	uint3 fi_xyz = make_uint3(floatingImage->nx, floatingImage->ny, floatingImage->nz);
	uint2 wi_tu = make_uint2(warpedImage->nt, warpedImage->nu);


	mat44ToCptr(*sourceIJKMatrix, sourceIJKMatrix_h);
	if (numMats)
		mat33ToCptr(jacMat, jacMat_h, numMats);

	char* floating = "floating";
	char* floating1 = "deformationFieldImage_d";
	char* floating2 = "warpedImage_d";
	char* floating3 = "mask_d";
	char* floating4 = "matrix";

	//printf("uploading %s\n", floating);

	//floatingImage_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&floatingImage_d), floatingImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(floatingImage_d, floatingImage->data, floatingImage->nvox * sizeof(float), hipMemcpyHostToDevice));

	//printf("uploading %s\n", floating1);
	//deformationFieldImage_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&deformationFieldImage_d), deformationFieldImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(deformationFieldImage_d, deformationFieldImage->data, deformationFieldImage->nvox * sizeof(float), hipMemcpyHostToDevice));

	//printf("uploading %s\n", floating2);
	//warpedImage_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&warpedImage_d), warpedImage->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(warpedImage_d, warpedImage->data, warpedImage->nvox * sizeof(float), hipMemcpyHostToDevice));

	//printf("uploading %s\n", floating3);
	//mask_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&mask_d), targetVoxelNumber * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(mask_d, mask, targetVoxelNumber * sizeof(int), hipMemcpyHostToDevice));

	//mask_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&dtiIndeces_d), 6 * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(dtiIndeces_d, dtiIndeces, 6 * sizeof(int), hipMemcpyHostToDevice));

	//printf("uploading %s\n", floating4);
	//sourceIJKMatrix_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&sourceIJKMatrix_d), 16 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(sourceIJKMatrix_d, sourceIJKMatrix_h, 16 * sizeof(float), hipMemcpyHostToDevice));

	//sourceIJKMatrix_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&jacMat_d), numMats * 9 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(jacMat_d, jacMat_h, numMats * 9 * sizeof(float), hipMemcpyHostToDevice));

	// The DTI are logged
	reg_dti_resampling_preprocessing<float>(floatingImage, &originalFloatingData, dtiIndeces);
	//reg_dti_resampling_preprocessing<float> << <mygrid, myblocks >> >(floatingImage_d, dtiIndeces, fi_xyz);

	//printf("kernel %s\n", floating);
	if (interp == 1)
		TrilinearResampleImage << <mygrid, myblocks >> >(floatingImage_d, deformationFieldImage_d, warpedImage_d, mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	else if (interp == 3)
		CubicSplineResampleImage3D << <mygrid, myblocks >> >(floatingImage_d, deformationFieldImage_d, warpedImage_d, mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	else
		NearestNeighborResampleImage << <mygrid, myblocks >> >(floatingImage_d, deformationFieldImage_d, warpedImage_d, mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	NR_CUDA_CHECK_KERNEL(mygrid, myblocks)

		//printf("copy %s\n", floating);
		NR_CUDA_SAFE_CALL(hipMemcpy(warpedImage->data, warpedImage_d, warpedImage->nvox * sizeof(float), hipMemcpyDeviceToHost));
	//printf("done %s\n", floating);
	// The temporary logged floating array is deleted
	if (originalFloatingData != NULL) {
		free(floatingImage->data);
		floatingImage->data = originalFloatingData;
		originalFloatingData = NULL;
	}
	// The interpolated tensors are reoriented and exponentiated
	//reg_dti_resampling_postprocessing<float> << <mygrid, myblocks >> >(warpedImage_d, NULL, mask_d, jacMat_d, dtiIndeces_d, fi_xyz, wi_tu);
	reg_dti_resampling_postprocessing<float>(warpedImage, mask, jacMat, dtiIndeces);

	hipFree(floatingImage_d);
	hipFree(deformationFieldImage_d);
	hipFree(warpedImage_d);
	hipFree(mask_d);
	hipFree(sourceIJKMatrix_d);
	hipFree(jacMat_d);



}

void runKernel2(nifti_image *floatingImage, nifti_image *warpedImage, int *mask, int interp, float paddingValue, int *dtiIndeces, mat33 * jacMat, float** floatingImage_d, float** warpedImage_d, float** deformationFieldImage_d,  int** mask_d) {


	long targetVoxelNumber = (long)warpedImage->nx*warpedImage->ny*warpedImage->nz;
	hipDeviceProp_t  prop;
	hipGetDeviceProperties(&prop, 0);
	unsigned int maxThreads = prop.maxThreadsDim[0];
	unsigned int maxBlocks = prop.maxThreadsDim[0];
	unsigned int blocks = (targetVoxelNumber % maxThreads) ? (targetVoxelNumber / maxThreads) + 1 : targetVoxelNumber / maxThreads;
	blocks = min1(blocks, maxBlocks);

	dim3 mygrid(blocks, 1, 1);
	dim3 myblocks(maxThreads, 1, 1);
	//printf("maxBlocks b: %d | maxThreads: %d\n", maxBlocks, maxThreads);
	//printf("blocks: %d | threads: %d\n", blocks, maxThreads);

	// The floating image data is copied in case one deal with DTI
	void *originalFloatingData = NULL;

	//number of jacobian matrices
	int numMats = 0;//needs to be transfered to a param 
	
	float *sourceIJKMatrix_h = (float*)malloc(16 * sizeof(float));
	float* jacMat_h = (float*)malloc(9 * numMats*sizeof(float));
	
	mat44 *sourceIJKMatrix;
	if (floatingImage->sform_code > 0)
		sourceIJKMatrix = &(floatingImage->sto_ijk);
	else sourceIJKMatrix = &(floatingImage->qto_ijk);

	float* sourceIJKMatrix_d, *jacMat_d;
	int* dtiIndeces_d;
	long2 voxelNumber = make_long2(warpedImage->nx*warpedImage->ny*warpedImage->nz, floatingImage->nx*floatingImage->ny*floatingImage->nz);
	uint3 fi_xyz = make_uint3(floatingImage->nx, floatingImage->ny, floatingImage->nz);
	uint2 wi_tu = make_uint2(warpedImage->nt, warpedImage->nu);


	mat44ToCptr(*sourceIJKMatrix, sourceIJKMatrix_h);
	if (numMats)
		mat33ToCptr(jacMat, jacMat_h, numMats);

	char* floating = "floating";
	char* floating1 = "deformationFieldImage_d";
	char* floating2 = "warpedImage_d";
	char* floating3 = "mask_d";
	char* floating4 = "matrix";

	//mask_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&dtiIndeces_d), 6 * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(dtiIndeces_d, dtiIndeces, 6 * sizeof(int), hipMemcpyHostToDevice));

	//sourceIJKMatrix_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&sourceIJKMatrix_d), 16 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(sourceIJKMatrix_d, sourceIJKMatrix_h, 16 * sizeof(float), hipMemcpyHostToDevice));

	//sourceIJKMatrix_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&jacMat_d), numMats * 9 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(jacMat_d, jacMat_h, numMats * 9 * sizeof(float), hipMemcpyHostToDevice));

	// The DTI are logged
	reg_dti_resampling_preprocessing<float>(floatingImage, &originalFloatingData, dtiIndeces);
	//reg_dti_resampling_preprocessing<float> << <mygrid, myblocks >> >(floatingImage_d, dtiIndeces, fi_xyz);

	if (interp == 1)
		TrilinearResampleImage << <mygrid, myblocks >> >(*floatingImage_d, *deformationFieldImage_d, *warpedImage_d, *mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	else if (interp == 3)
		CubicSplineResampleImage3D << <mygrid, myblocks >> >(*floatingImage_d, *deformationFieldImage_d, *warpedImage_d, *mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	else
		NearestNeighborResampleImage << <mygrid, myblocks >> >(*floatingImage_d, *deformationFieldImage_d, *warpedImage_d, *mask_d, sourceIJKMatrix_d, voxelNumber, fi_xyz, wi_tu, paddingValue);
	//NR_CUDA_CHECK_KERNEL(mygrid, myblocks)
	NR_CUDA_SAFE_CALL(hipDeviceSynchronize());

	//NR_CUDA_SAFE_CALL(hipMemcpy(warpedImage->data, *warpedImage_d, warpedImage->nvox * sizeof(float), hipMemcpyDeviceToHost));
	// The temporary logged floating array is deleted
	if (originalFloatingData != NULL) {
		free(floatingImage->data);
		floatingImage->data = originalFloatingData;
		originalFloatingData = NULL;
	}
	// The interpolated tensors are reoriented and exponentiated
	//reg_dti_resampling_postprocessing<float> << <mygrid, myblocks >> >(warpedImage_d, NULL, mask_d, jacMat_d, dtiIndeces_d, fi_xyz, wi_tu);
	reg_dti_resampling_postprocessing<float>(warpedImage, mask, jacMat, dtiIndeces);

	hipFree(sourceIJKMatrix_d);
	hipFree(jacMat_d);
	hipFree(dtiIndeces_d);
	
	//free(originalFloatingData);
	free(sourceIJKMatrix_h);
	free(jacMat_h);


}

void launchBlockMatching(nifti_image * target, nifti_image * result, _reg_blockMatchingParam *params, int *mask){

	float *targetImageArray_d;
	float *resultImageArray_d;
	float *targetPosition_d;
	float *resultPosition_d;
	int *activeBlock_d, *mask_d;

	//targetImageArray_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&targetImageArray_d), target->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(targetImageArray_d, target->data, target->nvox * sizeof(float), hipMemcpyHostToDevice));

	//resultImageArray_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&resultImageArray_d), result->nvox * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(resultImageArray_d, result->data, result->nvox * sizeof(float), hipMemcpyHostToDevice));

	//targetPosition_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&targetPosition_d), params->activeBlockNumber * 3 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(targetPosition_d, params->targetPosition, params->activeBlockNumber * 3 * sizeof(float), hipMemcpyHostToDevice));

	//resultPosition_d
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&resultPosition_d), params->activeBlockNumber * 3 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMemcpy(resultPosition_d, params->resultPosition, params->activeBlockNumber * 3 * sizeof(float), hipMemcpyHostToDevice));

	//activeBlock_d

	int3 bDim = make_int3(params->blockNumber[0], params->blockNumber[1], params->blockNumber[2]);
	const int numBlocks = bDim.x*bDim.y*bDim.z;
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&activeBlock_d), numBlocks  * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(activeBlock_d, params->activeBlock, numBlocks  * sizeof(int), hipMemcpyHostToDevice));
	
	NR_CUDA_SAFE_CALL(hipMalloc((void**)(&mask_d), target->nvox * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMemcpy(mask_d, mask, target->nvox * sizeof(int), hipMemcpyHostToDevice));

	block_matching_method_gpu(target, result, params, &targetImageArray_d, &resultImageArray_d, &targetPosition_d, &resultPosition_d, &activeBlock_d, &mask_d);

	
	//hipDeviceReset();
	/*hipFree(targetImageArray_d);
	hipFree(resultImageArray_d);
	hipFree(targetPosition_d);
	hipFree(resultPosition_d);
	hipFree(activeBlock_d);*/
}

void identityConst(){
	float* mat_h = (float*)malloc(16*sizeof(float));
	mat44* final;
	// Set the current transformation to identity
	final->m[0][0] = final->m[1][1] = final->m[2][2] = final->m[3][3] = 1.0f;
	final->m[0][1] = final->m[0][2] = final->m[0][3] = 0.0f;
	final->m[1][0] = final->m[1][2] = final->m[1][3] = 0.0f;
	final->m[2][0] = final->m[2][1] = final->m[2][3] = 0.0f;
	final->m[3][0] = final->m[3][1] = final->m[3][2] = 0.0f;
	mat44ToCptr(*final, mat_h);
	hipMemcpyToSymbol(HIP_SYMBOL(cIdentity), &mat_h, 16*sizeof(float));
}

void launchBlockMatching2(nifti_image * target,  _reg_blockMatchingParam *params, float **targetImageArray_d,
	float **resultImageArray_d,
	float **targetPosition_d,
	float **resultPosition_d,
	int **activeBlock_d, int **mask_d){



	block_matching_method_gpu3(target, params, targetImageArray_d, resultImageArray_d, targetPosition_d, resultPosition_d, activeBlock_d, mask_d);
}



void launchOptimizeAffine(_reg_blockMatchingParam* params, mat44* final, bool affine){

	//

	////    const unsigned num_points = params->activeBlockNumber;
	//const unsigned num_points = params->definedActiveBlock;
	//unsigned long num_equations = num_points * 3;
	//std::multimap<double, _reg_sorted_point3D> queue;
	//std::vector<_reg_sorted_point3D> top_points;
	//double distance = 0.0;
	//double lastDistance = std::numeric_limits<double>::max();
	//unsigned long i;

	//float* a_h, *w_h, *v_h, *r_h = (float*)malloc(num_equations*12*sizeof(float));
	//float* b_h = (float*)malloc(num_equations * sizeof(float));


	//// massive left hand side matrix
	//float ** a = new float *[num_equations];
	//for (unsigned k = 0; k < num_equations; ++k)
	//{
	//	a[k] = new float[12]; // full affine
	//}

	//// The array of singular values returned by svd
	//float *w = new float[12];

	//// v will be n x n
	//float **v = new float *[12];
	//for (unsigned k = 0; k < 12; ++k)
	//{
	//	v[k] = new float[12];
	//}

	//// Allocate memory for pseudoinverse
	//float **r = new float *[12];
	//for (unsigned k = 0; k < 12; ++k)
	//{
	//	r[k] = new float[num_equations];
	//}

	//// Allocate memory for RHS vector
	//float *b = new float[num_equations];

	//// The initial vector with all the input points
	//for (unsigned j = 0; j < num_points * 3; j += 3)
	//{
	//	top_points.push_back(_reg_sorted_point3D(&(params->targetPosition[j]), &(params->resultPosition[j]), 0.0f));
	//}

	//// estimate the optimal transformation while considering all the points
	//estimate_affine_transformation3D(top_points, final, a, w, v, r, b);

	//// Delete a, b and r. w and v will not change size in subsequent svd operations.
	//for (unsigned int k = 0; k < num_equations; ++k)
	//{
	//	delete[] a[k];
	//}
	//delete[] a;
	//delete[] b;

	//for (unsigned k = 0; k < 12; ++k)
	//{
	//	delete[] r[k];
	//}
	//delete[] r;


	//// The LS in the iterations is done on subsample of the input data
	//float * newResultPosition = new float[num_points * 3];
	//const unsigned long num_to_keep = (unsigned long)(num_points * (params->percent_to_keep / 100.0f));
	//num_equations = num_to_keep * 3;

	//// The LHS matrix
	//a = new float *[num_equations];
	//for (unsigned k = 0; k < num_equations; ++k)
	//{
	//	a[k] = new float[12]; // full affine
	//}

	//// Allocate memory for pseudoinverse
	//r = new float *[12];
	//for (unsigned k = 0; k < 12; ++k)
	//{
	//	r[k] = new float[num_equations];
	//}

	//// Allocate memory for RHS vector
	//b = new float[num_equations];
	//mat44 lastTransformation;
	//memset(&lastTransformation, 0, sizeof(mat44));

	//for (unsigned count = 0; count < MAX_ITERATIONS; ++count)
	//{
	//	// Transform the points in the target
	//	for (unsigned j = 0; j < num_points * 3; j += 3)
	//	{
	//		reg_mat44_mul(final, &(params->targetPosition[j]), &newResultPosition[j]);
	//	}

	//	queue = std::multimap<double, _reg_sorted_point3D>();
	//	for (unsigned j = 0; j < num_points * 3; j += 3)
	//	{
	//		distance = get_square_distance(&newResultPosition[j], &(params->resultPosition[j]));
	//		queue.insert(std::pair<double, _reg_sorted_point3D>(distance, _reg_sorted_point3D(&(params->targetPosition[j]),
	//			&(params->resultPosition[j]), distance)));
	//	}

	//	distance = 0.0;
	//	i = 0;
	//	top_points.clear();

	//	for (std::multimap<double, _reg_sorted_point3D>::iterator it = queue.begin();
	//		it != queue.end(); ++it, ++i)
	//	{
	//		if (i >= num_to_keep) break;
	//		top_points.push_back((*it).second);
	//		distance += (*it).first;
	//	}

	//	// If the change is not substantial or we are getting worst, we return
	//	if ((distance >= lastDistance) || (lastDistance - distance) < TOLERANCE)
	//	{
	//		// restore the last transformation
	//		copy_transformation_4x4(lastTransformation, *(final));
	//		break;
	//	}
	//	lastDistance = distance;
	//	copy_transformation_4x4(*(final), lastTransformation);
	//	estimate_affine_transformation3D(top_points, final, a, w, v, r, b);
	//}
	//delete[] newResultPosition;
	//delete[] b;
	//for (unsigned k = 0; k < 12; ++k)
	//{
	//	delete[] r[k];
	//}
	//delete[] r;

	//// free the memory
	//for (unsigned int k = 0; k < num_equations; ++k)
	//{
	//	delete[] a[k];
	//}
	//delete[] a;

	//delete[] w;
	//for (int k = 0; k < 12; ++k)
	//{
	//	delete[] v[k];
	//}
	//delete[] v;

}
void launchOptimizeRigid(_reg_blockMatchingParam* params, mat44* transformation_matrix, bool affine){}
